#include "hip/hip_runtime.h"
#include <chrono>
#include <stdio.h>

#define GRID_DIM 39063
#define BLOCK_DIM 256

using namespace std;

/*

        base = Array.pnew(dimensions: DIMS) do |indices|
            (indices[2]) % 133777
        end

        return Ikra::Symbolic.host_section(base) do |x|
            y = x
            old_data = x.__call__.to_command

            for r in 0...200
                if r % 2 == 0
                    if r % 3 == 0
                        y = y.pmap(with_index: true) do |i, indices|
                            (i + indices[3]) % 77689
                        end
                    else
                        y = y.pmap(with_index: true) do |i, indices|
                            (i + indices[0]) % 11799
                        end
                    end
                else
                    y = y.pmap(with_index: true) do |i, indices|
                        (i + indices[2]) % 1337
                    end

                    y = y.pmap(with_index: true) do |i, indices|
                        (i + indices[2]) % 8888888
                    end
                end

                y = y.pmap(with_index: true) do |i, indices|
                    (i + indices[2]) % 6678
                end

                old_data.free_memory
                old_data = y
            end

            y
        end

*/

__global__ void kernel_new(int *data) {
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 10000000) return;

    int idx_0 =_tid_ / 500000;
    int idx_1 = (_tid_ / 1000) % 500;
    int idx_2 = (_tid_ / 2) % 500;
    int idx_3 = (_tid_ / 1) % 2;

    // int indices[] = {idx_0, idx_1, idx_2, idx_3};

    data[_tid_] = idx_2 % 133777;
}

__global__ void kernel_1(int *data)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 10000000) return;

    int idx_0 =_tid_ / 500000;
    int idx_1 = (_tid_ / 1000) % 500;
    int idx_2 = (_tid_ / 2) % 500;
    int idx_3 = (_tid_ / 1) % 2;

    data[_tid_] = (data[_tid_] + idx_3) % 77689;
}

__global__ void kernel_2(int *data)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 10000000) return;

    int idx_0 =_tid_ / 500000;
    int idx_1 = (_tid_ / 1000) % 500;
    int idx_2 = (_tid_ / 2) % 500;
    int idx_3 = (_tid_ / 1) % 2;

    data[_tid_] = (data[_tid_] + idx_0) % 11799;
}

__global__ void kernel_3(int *data)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 10000000) return;

    int idx_0 =_tid_ / 500000;
    int idx_1 = (_tid_ / 1000) % 500;
    int idx_2 = (_tid_ / 2) % 500;
    int idx_3 = (_tid_ / 1) % 2;

    data[_tid_] = (data[_tid_] + idx_2) % 1337;
}

__global__ void kernel_4(int *data)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 10000000) return;

    int idx_0 =_tid_ / 500000;
    int idx_1 = (_tid_ / 1000) % 500;
    int idx_2 = (_tid_ / 2) % 500;
    int idx_3 = (_tid_ / 1) % 2;

    data[_tid_] = (data[_tid_] + idx_2) % 8888888;
}

__global__ void kernel_5(int *new_data, int *data)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 10000000) return;

    int idx_0 =_tid_ / 500000;
    int idx_1 = (_tid_ / 1000) % 500;
    int idx_2 = (_tid_ / 2) % 500;
    int idx_3 = (_tid_ / 1) % 2;

    new_data[_tid_] = (data[_tid_] + idx_2) % 6678;
}

int main()
{
    auto start_entire = chrono::high_resolution_clock::now();

    // Init
    hipDeviceSynchronize();

    long time_kernel = 0;
    long time_alloc = 0;
    long time_free = 0;
    long time_transfer = 0;

    // Measure kernel invocation
    auto start_time = chrono::high_resolution_clock::now();
    auto end_time = chrono::high_resolution_clock::now();
    long loop_time_elapsed;

    printf("START\n");
    int * data;
    hipMalloc(&data, (sizeof(int) * 10000000));
    hipDeviceSynchronize();

    end_time = chrono::high_resolution_clock::now();
    time_alloc += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

 
    start_time = chrono::high_resolution_clock::now();
    kernel_new<<<GRID_DIM, BLOCK_DIM>>>(data);
    hipDeviceSynchronize();
    end_time = chrono::high_resolution_clock::now();
    time_kernel += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

    for (int r = 0; r < 200; r++)
    {
        start_time = chrono::high_resolution_clock::now();
        if (r % 2 == 0) {
            if (r % 3 == 0) {
                kernel_1<<<GRID_DIM, BLOCK_DIM>>>(data);
            }
            else {
                kernel_2<<<GRID_DIM, BLOCK_DIM>>>(data);
            }
        } else {
            kernel_3<<<GRID_DIM, BLOCK_DIM>>>(data);
            kernel_4<<<GRID_DIM, BLOCK_DIM>>>(data);
        }
        hipDeviceSynchronize();

        end_time = chrono::high_resolution_clock::now();
        time_kernel += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

        start_time = chrono::high_resolution_clock::now();
        int * new_data;
        hipMalloc(&new_data, (sizeof(int) * 10000000));
        hipDeviceSynchronize();
        end_time = chrono::high_resolution_clock::now();
        time_alloc += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

        start_time = chrono::high_resolution_clock::now();
        kernel_5<<<GRID_DIM, BLOCK_DIM>>>(new_data, data);
        hipDeviceSynchronize();
        end_time = chrono::high_resolution_clock::now();
        time_kernel += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

        start_time = chrono::high_resolution_clock::now();
        hipFree(data);
        hipDeviceSynchronize();
        end_time = chrono::high_resolution_clock::now();
        time_free += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

        data = new_data;
    }

    hipDeviceSynchronize();

    // Copy back
    start_time = chrono::high_resolution_clock::now();
    int * tmp_result = (int *) malloc(sizeof(int) * 10000000);
    hipMemcpy(tmp_result, data, sizeof(int) * 10000000, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    end_time = chrono::high_resolution_clock::now();
    time_transfer += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();


    end_time = chrono::high_resolution_clock::now();
    int time_entire = chrono::duration_cast<chrono::microseconds>(end_time - start_entire).count();

    printf("alloc: %f\n", time_alloc / 1000.0);
    printf("kernel: %f\n", time_kernel / 1000.0);
    printf("transfer: %f\n", time_transfer / 1000.0f);
    printf("free: %f\n", time_free / 1000.f);
    printf("rest: %f\n", (time_entire - time_alloc - time_kernel - time_transfer - time_free) / 1000.0f);

    printf("END\n");  
}
