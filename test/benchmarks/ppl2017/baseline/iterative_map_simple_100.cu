#include "hip/hip_runtime.h"
#include <chrono>
#include <stdio.h>

#define GRID_DIM 39063
#define BLOCK_DIM 256

using namespace std;

__global__ void kernel_new(int *data) {
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 10000000) return;

    int idx_0 =_tid_ / 500000;
    int idx_1 = (_tid_ / 1000) % 500;
    int idx_2 = (_tid_ / 2) % 500;
    int idx_3 = (_tid_ / 1) % 2;

    // int indices[] = {idx_0, idx_1, idx_2, idx_3};

    data[_tid_] = idx_2 % 133777;
}

__global__ void kernel_5(int *new_data, int *data)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ >= 10000000) return;

    int idx_2 = (_tid_ / 2) % 500;

    new_data[_tid_] = (data[_tid_] + idx_2) % 13377;
}

int main()
{
    auto start_entire = chrono::high_resolution_clock::now();

    // Init
    hipDeviceSynchronize();

    long time_kernel = 0;
    long time_alloc = 0;
    long time_free = 0;
    long time_transfer = 0;

    // Measure kernel invocation
    auto start_time = chrono::high_resolution_clock::now();
    auto end_time = chrono::high_resolution_clock::now();
    long loop_time_elapsed;

    printf("START\n");
    int * data;
    hipMalloc(&data, (sizeof(int) * 10000000));
    hipDeviceSynchronize();

    end_time = chrono::high_resolution_clock::now();
    time_alloc += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

 
    start_time = chrono::high_resolution_clock::now();
    kernel_new<<<GRID_DIM, BLOCK_DIM>>>(data);
    hipDeviceSynchronize();
    end_time = chrono::high_resolution_clock::now();
    time_kernel += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

    for (int r = 0; r < 500; r++)
    {
        start_time = chrono::high_resolution_clock::now();
        int * new_data;
        hipMalloc(&new_data, (sizeof(int) * 10000000));
        hipDeviceSynchronize();
        end_time = chrono::high_resolution_clock::now();
        time_alloc += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

        start_time = chrono::high_resolution_clock::now();
        kernel_5<<<GRID_DIM, BLOCK_DIM>>>(new_data, data);
        hipDeviceSynchronize();
        end_time = chrono::high_resolution_clock::now();
        time_kernel += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

        start_time = chrono::high_resolution_clock::now();
        hipFree(data);
        hipDeviceSynchronize();
        end_time = chrono::high_resolution_clock::now();
        time_free += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

        data = new_data;
    }

    hipDeviceSynchronize();

    // Copy back
    start_time = chrono::high_resolution_clock::now();
    int * tmp_result = (int *) malloc(sizeof(int) * 10000000);
    hipMemcpy(tmp_result, data, sizeof(int) * 10000000, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    end_time = chrono::high_resolution_clock::now();
    time_transfer += chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();


    end_time = chrono::high_resolution_clock::now();
    int time_entire = chrono::duration_cast<chrono::microseconds>(end_time - start_entire).count();

    printf("alloc: %f\n", time_alloc / 1000.0);
    printf("kernel: %f\n", time_kernel / 1000.0);
    printf("transfer: %f\n", time_transfer / 1000.0f);
    printf("free: %f\n", time_free / 1000.f);
    printf("rest: %f\n", (time_entire - time_alloc - time_kernel - time_transfer - time_free) / 1000.0f);

    printf("END\n");  
}
