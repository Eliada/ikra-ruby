#include "hip/hip_runtime.h"
__global__ void main_kernel(float magnify, int hx_res, int hy_res, int iter_max, int* _result_)
{
    int i = blockIdx.x;
    int hx = (i % hx_res);
    int hy = (i / hx_res);
    float cx = ((((((float) (hx) / (float) (hx_res)) - 0.5) / magnify) * 3.0) - 0.7);
    float cy = (((((float) (hy) / (float) (hy_res)) - 0.5) / magnify) * 3.0);
    float x = 0.0;
    float y = 0.0;
    for (i = 0; i <= iter_max; i++)
    {
        float xx = (((x * x) - (y * y)) + cx);
        y = (((2.0 * x) * y) + cy);
        x = xx;
        if ((((x * x) + (y * y)) > 100))
        {
            i = 101;
            break;
        }
        ;
    }
    ;
    if ((i == 101))
    {
        ;
        _result_[blockIdx.x] = 1;
    }
    else
    {
        ;
        _result_[blockIdx.x] = 0;
    }
    ;
}
extern "C" __declspec(dllexport) int launch_kernel(float magnify, int hx_res, int hy_res, int iter_max)
{
    int * host_result = (int*) malloc(sizeof(int) * 250000);
    int * device_result;

    hipMalloc(&device_result, sizeof(int) * 250000);

    dim3 dim_grid(250000, 1, 1);
    dim3 dim_block(1, 1, 1);
    main_kernel<<<dim_grid, dim_block>>>(magnify, hx_res, hy_res, iter_max, device_result);

    hipDeviceSynchronize();
    hipMemcpy(host_result, device_result, sizeof(int) * 250000, hipMemcpyDeviceToHost);
    hipFree(device_result);

    return 1;
}
