#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <chrono>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

using namespace std;


/* ----- BEGIN Shared Library Export ----- */
// taken from http://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux

#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT __declspec(dllexport)
    #define IMPORT __declspec(dllimport)
#elif defined(_GCC)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif
/* ----- END Shared Library Export ----- */

/* ----- BEGIN Class Type ----- */
typedef int obj_id_t;
typedef int class_id_t;
/* ----- END Class Type ----- */

/* ----- BEGIN Environment (lexical variables) ----- */
// environment_struct must be defined later
typedef struct environment_struct environment_t;
/* ----- END Environment (lexical variables) ----- */


/* ----- BEGIN Forward declarations ----- */
typedef struct result_t result_t;
/* ----- END Forward declarations ----- */

// Define program result variable. Also contains benchmark numbers.
result_t *program_result;

// Variables for measuring time
chrono::high_resolution_clock::time_point start_time;
chrono::high_resolution_clock::time_point end_time;

/* ----- BEGIN Macros ----- */
#define timeStartMeasure() start_time = chrono::high_resolution_clock::now();

#define timeReportMeasure(result_var, variable_name) \
end_time = chrono::high_resolution_clock::now(); \
result_var->time_##variable_name = result_var->time_##variable_name + chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();
/* ----- END Macros ----- */
struct indexed_struct_4_lt_int_int_int_int_gt_t
{
    int field_0;
int field_1;
int field_2;
int field_3;
};

/* ----- BEGIN Structs ----- */
struct variable_size_array_t {
    void *content;
    int size;

    variable_size_array_t(void *content_ = NULL, int size_ = 0) : content(content_), size(size_) { }; 

    static const variable_size_array_t error_return_value;
};

// error_return_value is used in case a host section terminates abnormally
const variable_size_array_t variable_size_array_t::error_return_value = 
    variable_size_array_t(NULL, 0);

/* ----- BEGIN Union Type ----- */
typedef union union_type_value {
    obj_id_t object_id;
    int int_;
    float float_;
    bool bool_;
    void *pointer;
    variable_size_array_t variable_size_array;

    __host__ __device__ union_type_value(int value) : int_(value) { };
    __host__ __device__ union_type_value(float value) : float_(value) { };
    __host__ __device__ union_type_value(bool value) : bool_(value) { };
    __host__ __device__ union_type_value(void *value) : pointer(value) { };
    __host__ __device__ union_type_value(variable_size_array_t value) : variable_size_array(value) { };

    __host__ __device__ static union_type_value from_object_id(obj_id_t value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_int(int value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_float(float value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_bool(bool value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_pointer(void *value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_variable_size_array_t(variable_size_array_t value)
    {
        return union_type_value(value);
    }
} union_v_t;

typedef struct union_type_struct
{
    class_id_t class_id;
    union_v_t value;

    __host__ __device__ union_type_struct(
        class_id_t class_id_ = 0, union_v_t value_ = union_v_t(0))
        : class_id(class_id_), value(value_) { };

    static const union_type_struct error_return_value;
} union_t;

// error_return_value is used in case a host section terminates abnormally
const union_type_struct union_t::error_return_value = union_type_struct(0, union_v_t(0));
/* ----- END Union Type ----- */

typedef struct result_t {
    variable_size_array_t result;
    int last_error;

    uint64_t time_setup_cuda;
    uint64_t time_prepare_env;
    uint64_t time_kernel;
    uint64_t time_free_memory;
    uint64_t time_transfer_memory;
    uint64_t time_allocate_memory;

    // Memory management
    vector<void*> *device_allocations;
} result_t;
/* ----- END Structs ----- */

struct array_command_1 {
    // Ikra::Symbolic::ArrayIndexCommand
    indexed_struct_4_lt_int_int_int_int_gt_t *result;
    __host__ __device__ array_command_1(indexed_struct_4_lt_int_int_int_int_gt_t *result = NULL) : result(result) { }
};
struct array_command_2 {
    // Ikra::Symbolic::ArrayCombineCommand
    int *result;
    array_command_1 *input_0;
    __host__ __device__ array_command_2(int *result = NULL, array_command_1 *input_0 = NULL) : result(result), input_0(input_0) { }
};
struct array_command_3 {
    // Ikra::Symbolic::FixedSizeArrayInHostSectionCommand
    int *result;
    variable_size_array_t input_0;
    __host__ __device__ array_command_3(int *result = NULL, variable_size_array_t input_0 = variable_size_array_t::error_return_value) : result(result), input_0(input_0) { }
    int size() { return input_0.size; }
};
struct array_command_5 {
    // Ikra::Symbolic::ArrayIndexCommand
    indexed_struct_4_lt_int_int_int_int_gt_t *result;
    __host__ __device__ array_command_5(indexed_struct_4_lt_int_int_int_int_gt_t *result = NULL) : result(result) { }
};
struct array_command_4 {
    // Ikra::Symbolic::ArrayStencilCommand
    int *result;
    array_command_3 *input_0;
    array_command_5 *input_1;
    __host__ __device__ array_command_4(int *result = NULL, array_command_3 *input_0 = NULL, array_command_5 *input_1 = NULL) : result(result), input_0(input_0), input_1(input_1) { }
};
struct environment_struct
{
};

// TODO: There should be a better to check if _block_k_2_ is already defined
#ifndef _block_k_2__func
#define _block_k_2__func
__device__ int _block_k_2_(environment_t *_env_, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    {
        return (((indices.field_0 + indices.field_1)) % ((((indices.field_3 + ({ int _temp_var_117 = ((indices.field_1 % 4));
        (_temp_var_117 == 0 ? indices.field_0 : (_temp_var_117 == 1 ? indices.field_1 : (_temp_var_117 == 2 ? indices.field_2 : (_temp_var_117 == 3 ? indices.field_3 : NULL)))); }))) + 7)));
    }
}

#endif


__global__ void kernel_197(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_2_(_env_, ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_2_ is already defined
#ifndef _block_k_2__func
#define _block_k_2__func
__device__ int _block_k_2_(environment_t *_env_, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    {
        return (((indices.field_0 + indices.field_1)) % ((((indices.field_3 + ({ int _temp_var_118 = ((indices.field_1 % 4));
        (_temp_var_118 == 0 ? indices.field_0 : (_temp_var_118 == 1 ? indices.field_1 : (_temp_var_118 == 2 ? indices.field_2 : (_temp_var_118 == 3 ? indices.field_3 : NULL)))); }))) + 7)));
    }
}

#endif


__global__ void kernel_199(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_2_(_env_, ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}




__global__ void kernel_201(environment_t *_env_, int _num_threads_, int *_result_, int *_array_203_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_203_[_tid_];
    }
}




__global__ void kernel_206(environment_t *_env_, int _num_threads_, int *_result_, int *_array_208_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_208_[_tid_];
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_121 = ((({ int _temp_var_122 = ((({ int _temp_var_123 = ((values[2] % 4));
        (_temp_var_123 == 0 ? indices.field_0 : (_temp_var_123 == 1 ? indices.field_1 : (_temp_var_123 == 2 ? indices.field_2 : (_temp_var_123 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_122 == 0 ? indices.field_0 : (_temp_var_122 == 1 ? indices.field_1 : (_temp_var_122 == 2 ? indices.field_2 : (_temp_var_122 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_121 == 0 ? indices.field_0 : (_temp_var_121 == 1 ? indices.field_1 : (_temp_var_121 == 2 ? indices.field_2 : (_temp_var_121 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_204(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_207)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_209;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_209 = _block_k_4_(_env_, _kernel_result_207[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_207[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_207[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_207[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_209 = 37;
    }
        
        _result_[_tid_] = temp_stencil_209;
    }
}




__global__ void kernel_210(environment_t *_env_, int _num_threads_, int *_result_, int *_array_212_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_212_[_tid_];
    }
}




__global__ void kernel_215(environment_t *_env_, int _num_threads_, int *_result_, int *_array_217_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_217_[_tid_];
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_126 = ((({ int _temp_var_127 = ((({ int _temp_var_128 = ((values[2] % 4));
        (_temp_var_128 == 0 ? indices.field_0 : (_temp_var_128 == 1 ? indices.field_1 : (_temp_var_128 == 2 ? indices.field_2 : (_temp_var_128 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_127 == 0 ? indices.field_0 : (_temp_var_127 == 1 ? indices.field_1 : (_temp_var_127 == 2 ? indices.field_2 : (_temp_var_127 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_126 == 0 ? indices.field_0 : (_temp_var_126 == 1 ? indices.field_1 : (_temp_var_126 == 2 ? indices.field_2 : (_temp_var_126 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_213(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_216)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_218;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_218 = _block_k_4_(_env_, _kernel_result_216[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_216[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_216[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_216[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_218 = 37;
    }
        
        _result_[_tid_] = temp_stencil_218;
    }
}




__global__ void kernel_219(environment_t *_env_, int _num_threads_, int *_result_, int *_array_221_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_221_[_tid_];
    }
}




__global__ void kernel_224(environment_t *_env_, int _num_threads_, int *_result_, int *_array_226_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_226_[_tid_];
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_131 = ((({ int _temp_var_132 = ((({ int _temp_var_133 = ((values[2] % 4));
        (_temp_var_133 == 0 ? indices.field_0 : (_temp_var_133 == 1 ? indices.field_1 : (_temp_var_133 == 2 ? indices.field_2 : (_temp_var_133 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_132 == 0 ? indices.field_0 : (_temp_var_132 == 1 ? indices.field_1 : (_temp_var_132 == 2 ? indices.field_2 : (_temp_var_132 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_131 == 0 ? indices.field_0 : (_temp_var_131 == 1 ? indices.field_1 : (_temp_var_131 == 2 ? indices.field_2 : (_temp_var_131 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_222(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_225)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_227;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_227 = _block_k_4_(_env_, _kernel_result_225[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_225[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_225[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_225[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_227 = 37;
    }
        
        _result_[_tid_] = temp_stencil_227;
    }
}




__global__ void kernel_228(environment_t *_env_, int _num_threads_, int *_result_, int *_array_230_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_230_[_tid_];
    }
}




__global__ void kernel_233(environment_t *_env_, int _num_threads_, int *_result_, int *_array_235_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_235_[_tid_];
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_136 = ((({ int _temp_var_137 = ((({ int _temp_var_138 = ((values[2] % 4));
        (_temp_var_138 == 0 ? indices.field_0 : (_temp_var_138 == 1 ? indices.field_1 : (_temp_var_138 == 2 ? indices.field_2 : (_temp_var_138 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_137 == 0 ? indices.field_0 : (_temp_var_137 == 1 ? indices.field_1 : (_temp_var_137 == 2 ? indices.field_2 : (_temp_var_137 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_136 == 0 ? indices.field_0 : (_temp_var_136 == 1 ? indices.field_1 : (_temp_var_136 == 2 ? indices.field_2 : (_temp_var_136 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_231(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_234)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_236;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_236 = _block_k_4_(_env_, _kernel_result_234[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_234[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_234[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_234[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_236 = 37;
    }
        
        _result_[_tid_] = temp_stencil_236;
    }
}




__global__ void kernel_237(environment_t *_env_, int _num_threads_, int *_result_, int *_array_239_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_239_[_tid_];
    }
}




__global__ void kernel_242(environment_t *_env_, int _num_threads_, int *_result_, int *_array_244_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_244_[_tid_];
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_143 = ((({ int _temp_var_144 = ((({ int _temp_var_145 = ((values[2] % 4));
        (_temp_var_145 == 0 ? indices.field_0 : (_temp_var_145 == 1 ? indices.field_1 : (_temp_var_145 == 2 ? indices.field_2 : (_temp_var_145 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_144 == 0 ? indices.field_0 : (_temp_var_144 == 1 ? indices.field_1 : (_temp_var_144 == 2 ? indices.field_2 : (_temp_var_144 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_143 == 0 ? indices.field_0 : (_temp_var_143 == 1 ? indices.field_1 : (_temp_var_143 == 2 ? indices.field_2 : (_temp_var_143 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_240(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_243)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_245;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_245 = _block_k_4_(_env_, _kernel_result_243[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_243[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_243[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_243[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_245 = 37;
    }
        
        _result_[_tid_] = temp_stencil_245;
    }
}


#undef checkErrorReturn
#define checkErrorReturn(result_var, expr) \
if (result_var->last_error = expr) \
{\
    hipError_t error = hipGetLastError();\
    printf("!!! Cuda Failure %s:%d (%i): '%s'\n", __FILE__, __LINE__, expr, hipGetErrorString(error));\
    hipDeviceReset();\
    return variable_size_array_t::error_return_value;\
}

variable_size_array_t _host_section__(environment_t *host_env, environment_t *dev_env, result_t *program_result)
{
    array_command_2 * x = new array_command_2();
    union_t _ssa_var_old_old_data_3;
    array_command_4 * _ssa_var_y_6;
    union_t _ssa_var_old_data_5;
    union_t _ssa_var_old_old_data_4;
    int r;
    union_t _ssa_var_old_data_2;
    union_t _ssa_var_y_1;
    {
        _ssa_var_y_1 = union_t(10, union_v_t::from_pointer((void *) new array_command_3(NULL, ({
            // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]
        
            array_command_2 * cmd = x;
        
            if (cmd->result == 0) {
                    timeStartMeasure();
            int * _kernel_result_198;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_198, (sizeof(int) * 10000000)));
            program_result->device_allocations->push_back(_kernel_result_198);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_197<<<39063, 256>>>(dev_env, 10000000, _kernel_result_198);
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);
                cmd->result = _kernel_result_198;
        
                
            }
        
            variable_size_array_t((void *) cmd->result, 10000000);
        }))));
        _ssa_var_old_data_2 = union_t(19, union_v_t::from_pointer((void *) x));
        _ssa_var_old_old_data_3 = union_t(19, union_v_t::from_pointer((void *) x));
        for (r = 0; r <= (200 - 1); r++)
        {
            _ssa_var_old_old_data_4 = _ssa_var_old_data_2;
            _ssa_var_old_data_5 = _ssa_var_y_1;
            _ssa_var_y_6 = new array_command_4(NULL, new array_command_3(NULL, ({
                variable_size_array_t _polytemp_result_119;
                {
                    union_t _polytemp_expr_120 = _ssa_var_y_1;
                    switch (_polytemp_expr_120.class_id)
                    {
                        case 10: /* [Ikra::Symbolic::FixedSizeArrayInHostSectionCommand, size = 10000000] (Ikra::Symbolic::ArrayCommand) */ _polytemp_result_119 = ({
                            // [Ikra::Symbolic::FixedSizeArrayInHostSectionCommand, size = 10000000]
                        
                            array_command_3 * cmd = (array_command_3 *) _polytemp_expr_120.value.pointer;
                        
                            if (cmd->result == 0) {
                                    timeStartMeasure();
                            int * _kernel_result_202;
                            checkErrorReturn(program_result, hipMalloc(&_kernel_result_202, (sizeof(int) * 10000000)));
                            program_result->device_allocations->push_back(_kernel_result_202);
                            timeReportMeasure(program_result, allocate_memory);
                            timeStartMeasure();
                            kernel_201<<<39063, 256>>>(dev_env, 10000000, _kernel_result_202, ((int *) cmd->input_0.content));
                            checkErrorReturn(program_result, hipPeekAtLastError());
                            checkErrorReturn(program_result, hipDeviceSynchronize());
                            timeReportMeasure(program_result, kernel);
                                cmd->result = _kernel_result_202;
                        
                                
                            }
                        
                            variable_size_array_t((void *) cmd->result, 10000000);
                        }); break;
                        case 20: /* [Ikra::Symbolic::ArrayStencilCommand, size = 10000000] (Ikra::Symbolic::ArrayCommand) */ _polytemp_result_119 = ({
                            // [Ikra::Symbolic::ArrayStencilCommand, size = 10000000]: [SendNode: [SendNode: [SendNode: [LVarReadNode: _ssa_var_y_1].__call__()].to_command()].pstencil([ArrayNode: [[ArrayNode: [<-1>, <0>, <0>, <0>]], [ArrayNode: [<0>, <0>, <0>, <0>]], [ArrayNode: [<1>, <0>, <0>, <0>]], [ArrayNode: [<-1>, <-1>, <0>, <0>]]]]; <37>; [HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
                        
                            array_command_4 * cmd = (array_command_4 *) _polytemp_expr_120.value.pointer;
                        
                            if (cmd->result == 0) {
                                    timeStartMeasure();
                            int * _kernel_result_207;
                            checkErrorReturn(program_result, hipMalloc(&_kernel_result_207, (sizeof(int) * 10000000)));
                            program_result->device_allocations->push_back(_kernel_result_207);
                            timeReportMeasure(program_result, allocate_memory);
                            timeStartMeasure();
                            kernel_206<<<39063, 256>>>(dev_env, 10000000, _kernel_result_207, ((int *) ((int *) cmd->input_0->input_0.content)));
                            checkErrorReturn(program_result, hipPeekAtLastError());
                            checkErrorReturn(program_result, hipDeviceSynchronize());
                            timeReportMeasure(program_result, kernel);    timeStartMeasure();
                            int * _kernel_result_205;
                            checkErrorReturn(program_result, hipMalloc(&_kernel_result_205, (sizeof(int) * 10000000)));
                            program_result->device_allocations->push_back(_kernel_result_205);
                            timeReportMeasure(program_result, allocate_memory);
                            timeStartMeasure();
                            kernel_204<<<39063, 256>>>(dev_env, 10000000, _kernel_result_205, _kernel_result_207);
                            checkErrorReturn(program_result, hipPeekAtLastError());
                            checkErrorReturn(program_result, hipDeviceSynchronize());
                            timeReportMeasure(program_result, kernel);
                                cmd->result = _kernel_result_205;
                        
                                    timeStartMeasure();
                        
                            if (_kernel_result_207 != cmd->result) {
                                // Don't free memory if it is the result. There is already a similar check in
                                // program_builder (free all except for last). However, this check is not sufficient in
                                // case the same array is reused!
                        
                                checkErrorReturn(program_result, hipFree(_kernel_result_207));
                                // Remove from list of allocations
                                program_result->device_allocations->erase(
                                    std::remove(
                                        program_result->device_allocations->begin(),
                                        program_result->device_allocations->end(),
                                        _kernel_result_207),
                                    program_result->device_allocations->end());
                            }
                        
                            timeReportMeasure(program_result, free_memory);
                        
                            }
                        
                            variable_size_array_t((void *) cmd->result, 10000000);
                        }); break;
                    }
                }
                _polytemp_result_119;
            })));
            if (((r > 1)))
            {
                ({
                    bool _polytemp_result_139;
                    {
                        union_t _polytemp_expr_140 = _ssa_var_old_old_data_4;
                        switch (_polytemp_expr_140.class_id)
                        {
                            case 19: /* [Ikra::Symbolic::ArrayCombineCommand, size = 10000000] (Ikra::Symbolic::ArrayCommand) */ _polytemp_result_139 = ({
                                array_command_2 * cmd_to_free = (array_command_2 *) _polytemp_expr_140.value.pointer;
                            
                                timeStartMeasure();
                                bool freed_memory = false;
                            
                                if (cmd_to_free->result != 0) {
                                    checkErrorReturn(program_result, hipFree(cmd_to_free->result));;
                            
                                    // Remove from list of allocations
                                    program_result->device_allocations->erase(
                                        std::remove(
                                            program_result->device_allocations->begin(),
                                            program_result->device_allocations->end(),
                                            cmd_to_free->result),
                                        program_result->device_allocations->end());
                            
                                    freed_memory = true;
                                }
                            
                                timeReportMeasure(program_result, free_memory);
                                
                                freed_memory;
                            }); break;
                            case 10: /* [Ikra::Symbolic::FixedSizeArrayInHostSectionCommand, size = 10000000] (Ikra::Symbolic::ArrayCommand) */ _polytemp_result_139 = ({
                                array_command_3 * cmd_to_free = (array_command_3 *) _polytemp_expr_140.value.pointer;
                            
                                timeStartMeasure();
                                bool freed_memory = false;
                            
                                if (cmd_to_free->result != 0) {
                                    checkErrorReturn(program_result, hipFree(cmd_to_free->result));;
                            
                                    // Remove from list of allocations
                                    program_result->device_allocations->erase(
                                        std::remove(
                                            program_result->device_allocations->begin(),
                                            program_result->device_allocations->end(),
                                            cmd_to_free->result),
                                        program_result->device_allocations->end());
                            
                                    freed_memory = true;
                                }
                            
                                timeReportMeasure(program_result, free_memory);
                                
                                freed_memory;
                            }); break;
                            case 20: /* [Ikra::Symbolic::ArrayStencilCommand, size = 10000000] (Ikra::Symbolic::ArrayCommand) */ _polytemp_result_139 = ({
                                array_command_4 * cmd_to_free = (array_command_4 *) _polytemp_expr_140.value.pointer;
                            
                                timeStartMeasure();
                                bool freed_memory = false;
                            
                                if (cmd_to_free->result != 0) {
                                    checkErrorReturn(program_result, hipFree(cmd_to_free->result));;
                            
                                    // Remove from list of allocations
                                    program_result->device_allocations->erase(
                                        std::remove(
                                            program_result->device_allocations->begin(),
                                            program_result->device_allocations->end(),
                                            cmd_to_free->result),
                                        program_result->device_allocations->end());
                            
                                    freed_memory = true;
                                }
                            
                                timeReportMeasure(program_result, free_memory);
                                
                                freed_memory;
                            }); break;
                        }
                    }
                    _polytemp_result_139;
                });
            }
            else
            {
            
            }
            _ssa_var_y_1 = union_t(20, union_v_t::from_pointer((void *) _ssa_var_y_6));
            _ssa_var_old_data_2 = _ssa_var_old_data_5;
            _ssa_var_old_old_data_3 = _ssa_var_old_old_data_4;
        }
        r--;
        return ({
            variable_size_array_t _polytemp_result_141;
            {
                union_t _polytemp_expr_142 = _ssa_var_y_1;
                switch (_polytemp_expr_142.class_id)
                {
                    case 10: /* [Ikra::Symbolic::FixedSizeArrayInHostSectionCommand, size = 10000000] (Ikra::Symbolic::ArrayCommand) */ _polytemp_result_141 = ({
                        // [Ikra::Symbolic::FixedSizeArrayInHostSectionCommand, size = 10000000]
                    
                        array_command_3 * cmd = (array_command_3 *) _polytemp_expr_142.value.pointer;
                    
                        if (cmd->result == 0) {
                                timeStartMeasure();
                        int * _kernel_result_238;
                        checkErrorReturn(program_result, hipMalloc(&_kernel_result_238, (sizeof(int) * 10000000)));
                        program_result->device_allocations->push_back(_kernel_result_238);
                        timeReportMeasure(program_result, allocate_memory);
                        timeStartMeasure();
                        kernel_237<<<39063, 256>>>(dev_env, 10000000, _kernel_result_238, ((int *) cmd->input_0.content));
                        checkErrorReturn(program_result, hipPeekAtLastError());
                        checkErrorReturn(program_result, hipDeviceSynchronize());
                        timeReportMeasure(program_result, kernel);
                            cmd->result = _kernel_result_238;
                    
                            
                        }
                    
                        variable_size_array_t((void *) cmd->result, 10000000);
                    }); break;
                    case 20: /* [Ikra::Symbolic::ArrayStencilCommand, size = 10000000] (Ikra::Symbolic::ArrayCommand) */ _polytemp_result_141 = ({
                        // [Ikra::Symbolic::ArrayStencilCommand, size = 10000000]: [SendNode: [SendNode: [SendNode: [LVarReadNode: _ssa_var_y_1].__call__()].to_command()].pstencil([ArrayNode: [[ArrayNode: [<-1>, <0>, <0>, <0>]], [ArrayNode: [<0>, <0>, <0>, <0>]], [ArrayNode: [<1>, <0>, <0>, <0>]], [ArrayNode: [<-1>, <-1>, <0>, <0>]]]]; <37>; [HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
                    
                        array_command_4 * cmd = (array_command_4 *) _polytemp_expr_142.value.pointer;
                    
                        if (cmd->result == 0) {
                                timeStartMeasure();
                        int * _kernel_result_243;
                        checkErrorReturn(program_result, hipMalloc(&_kernel_result_243, (sizeof(int) * 10000000)));
                        program_result->device_allocations->push_back(_kernel_result_243);
                        timeReportMeasure(program_result, allocate_memory);
                        timeStartMeasure();
                        kernel_242<<<39063, 256>>>(dev_env, 10000000, _kernel_result_243, ((int *) ((int *) cmd->input_0->input_0.content)));
                        checkErrorReturn(program_result, hipPeekAtLastError());
                        checkErrorReturn(program_result, hipDeviceSynchronize());
                        timeReportMeasure(program_result, kernel);    timeStartMeasure();
                        int * _kernel_result_241;
                        checkErrorReturn(program_result, hipMalloc(&_kernel_result_241, (sizeof(int) * 10000000)));
                        program_result->device_allocations->push_back(_kernel_result_241);
                        timeReportMeasure(program_result, allocate_memory);
                        timeStartMeasure();
                        kernel_240<<<39063, 256>>>(dev_env, 10000000, _kernel_result_241, _kernel_result_243);
                        checkErrorReturn(program_result, hipPeekAtLastError());
                        checkErrorReturn(program_result, hipDeviceSynchronize());
                        timeReportMeasure(program_result, kernel);
                            cmd->result = _kernel_result_241;
                    
                                timeStartMeasure();
                    
                        if (_kernel_result_243 != cmd->result) {
                            // Don't free memory if it is the result. There is already a similar check in
                            // program_builder (free all except for last). However, this check is not sufficient in
                            // case the same array is reused!
                    
                            checkErrorReturn(program_result, hipFree(_kernel_result_243));
                            // Remove from list of allocations
                            program_result->device_allocations->erase(
                                std::remove(
                                    program_result->device_allocations->begin(),
                                    program_result->device_allocations->end(),
                                    _kernel_result_243),
                                program_result->device_allocations->end());
                        }
                    
                        timeReportMeasure(program_result, free_memory);
                    
                        }
                    
                        variable_size_array_t((void *) cmd->result, 10000000);
                    }); break;
                }
            }
            _polytemp_result_141;
        });
    }
}

#undef checkErrorReturn
#define checkErrorReturn(result_var, expr) \
expr

extern "C" EXPORT result_t *launch_kernel(environment_t *host_env)
{
    // CUDA Initialization
    program_result = new result_t();
    program_result->device_allocations = new vector<void*>();

    timeStartMeasure();

    hipError_t cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        program_result->last_error = -1;
        return program_result;
    }

    checkErrorReturn(program_result, hipFree(0));

    timeReportMeasure(program_result, setup_cuda);


    /* Prepare environment */
        /* Allocate device environment and copy over struct */
    environment_t *dev_env;

    timeStartMeasure();
    checkErrorReturn(program_result, hipMalloc(&dev_env, sizeof(environment_t)));
    timeReportMeasure(program_result, allocate_memory);

    timeStartMeasure();
    checkErrorReturn(program_result, hipMemcpy(dev_env, host_env, sizeof(environment_t), hipMemcpyHostToDevice));
    timeReportMeasure(program_result, transfer_memory);
    


    /* Copy back memory and set pointer of result */
    program_result->result = ({
    variable_size_array_t device_array = _host_section__(host_env, dev_env, program_result);
    int * tmp_result = (int *) malloc(sizeof(int) * device_array.size);

    timeStartMeasure();
    checkErrorReturn(program_result, hipMemcpy(tmp_result, device_array.content, sizeof(int) * device_array.size, hipMemcpyDeviceToHost));
    timeReportMeasure(program_result, transfer_memory);

    variable_size_array_t((void *) tmp_result, device_array.size);
});

    /* Free device memory */
    timeStartMeasure();

    for (
        auto device_ptr = program_result->device_allocations->begin(); 
        device_ptr < program_result->device_allocations->end(); 
        device_ptr++)
    {
        checkErrorReturn(program_result, hipFree(*device_ptr));
    }

    delete program_result->device_allocations;

    timeReportMeasure(program_result, free_memory);

    return program_result;
}
