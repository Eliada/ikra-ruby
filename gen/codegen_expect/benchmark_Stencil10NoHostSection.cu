#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <chrono>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

using namespace std;


/* ----- BEGIN Shared Library Export ----- */
// taken from http://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux

#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT __declspec(dllexport)
    #define IMPORT __declspec(dllimport)
#elif defined(_GCC)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif
/* ----- END Shared Library Export ----- */

/* ----- BEGIN Class Type ----- */
typedef int obj_id_t;
typedef int class_id_t;
/* ----- END Class Type ----- */

/* ----- BEGIN Environment (lexical variables) ----- */
// environment_struct must be defined later
typedef struct environment_struct environment_t;
/* ----- END Environment (lexical variables) ----- */


/* ----- BEGIN Forward declarations ----- */
typedef struct result_t result_t;
/* ----- END Forward declarations ----- */

// Define program result variable. Also contains benchmark numbers.
result_t *program_result;

// Variables for measuring time
chrono::high_resolution_clock::time_point start_time;
chrono::high_resolution_clock::time_point end_time;

/* ----- BEGIN Macros ----- */
#define timeStartMeasure() start_time = chrono::high_resolution_clock::now();

#define timeReportMeasure(result_var, variable_name) \
end_time = chrono::high_resolution_clock::now(); \
result_var->time_##variable_name = result_var->time_##variable_name + chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();
/* ----- END Macros ----- */
struct indexed_struct_4_lt_int_int_int_int_gt_t
{
    int field_0;
int field_1;
int field_2;
int field_3;
};

/* ----- BEGIN Structs ----- */
struct variable_size_array_t {
    void *content;
    int size;

    variable_size_array_t(void *content_ = NULL, int size_ = 0) : content(content_), size(size_) { }; 

    static const variable_size_array_t error_return_value;
};

// error_return_value is used in case a host section terminates abnormally
const variable_size_array_t variable_size_array_t::error_return_value = 
    variable_size_array_t(NULL, 0);

/* ----- BEGIN Union Type ----- */
typedef union union_type_value {
    obj_id_t object_id;
    int int_;
    float float_;
    bool bool_;
    void *pointer;
    variable_size_array_t variable_size_array;

    __host__ __device__ union_type_value(int value) : int_(value) { };
    __host__ __device__ union_type_value(float value) : float_(value) { };
    __host__ __device__ union_type_value(bool value) : bool_(value) { };
    __host__ __device__ union_type_value(void *value) : pointer(value) { };
    __host__ __device__ union_type_value(variable_size_array_t value) : variable_size_array(value) { };

    __host__ __device__ static union_type_value from_object_id(obj_id_t value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_int(int value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_float(float value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_bool(bool value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_pointer(void *value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_variable_size_array_t(variable_size_array_t value)
    {
        return union_type_value(value);
    }
} union_v_t;

typedef struct union_type_struct
{
    class_id_t class_id;
    union_v_t value;

    __host__ __device__ union_type_struct(
        class_id_t class_id_ = 0, union_v_t value_ = union_v_t(0))
        : class_id(class_id_), value(value_) { };

    static const union_type_struct error_return_value;
} union_t;

// error_return_value is used in case a host section terminates abnormally
const union_type_struct union_t::error_return_value = union_type_struct(0, union_v_t(0));
/* ----- END Union Type ----- */

typedef struct result_t {
    variable_size_array_t result;
    int last_error;

    uint64_t time_setup_cuda;
    uint64_t time_prepare_env;
    uint64_t time_kernel;
    uint64_t time_free_memory;
    uint64_t time_transfer_memory;
    uint64_t time_allocate_memory;

    // Memory management
    vector<void*> *device_allocations;
} result_t;
/* ----- END Structs ----- */


struct environment_struct
{
};

// TODO: There should be a better to check if _block_k_2_ is already defined
#ifndef _block_k_2__func
#define _block_k_2__func
__device__ int _block_k_2_(environment_t *_env_, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    {
        return (((indices.field_0 + indices.field_1)) % ((((indices.field_3 + ({ int _temp_var_1 = ((indices.field_1 % 4));
        (_temp_var_1 == 0 ? indices.field_0 : (_temp_var_1 == 1 ? indices.field_1 : (_temp_var_1 == 2 ? indices.field_2 : (_temp_var_1 == 3 ? indices.field_3 : NULL)))); }))) + 7)));
    }
}

#endif


__global__ void kernel_21(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_2_(_env_, ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_3_ is already defined
#ifndef _block_k_3__func
#define _block_k_3__func
__device__ int _block_k_3_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_2 = ((({ int _temp_var_3 = ((({ int _temp_var_4 = ((values[2] % 4));
        (_temp_var_4 == 0 ? indices.field_0 : (_temp_var_4 == 1 ? indices.field_1 : (_temp_var_4 == 2 ? indices.field_2 : (_temp_var_4 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_3 == 0 ? indices.field_0 : (_temp_var_3 == 1 ? indices.field_1 : (_temp_var_3 == 2 ? indices.field_2 : (_temp_var_3 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_2 == 0 ? indices.field_0 : (_temp_var_2 == 1 ? indices.field_1 : (_temp_var_2 == 2 ? indices.field_2 : (_temp_var_2 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_19(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_22)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_23;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_23 = _block_k_3_(_env_, _kernel_result_22[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_22[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_22[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_22[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_23 = 37;
    }
        
        _result_[_tid_] = temp_stencil_23;
    }
}



// TODO: There should be a better to check if _block_k_5_ is already defined
#ifndef _block_k_5__func
#define _block_k_5__func
__device__ int _block_k_5_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_5 = ((({ int _temp_var_6 = ((({ int _temp_var_7 = ((values[2] % 4));
        (_temp_var_7 == 0 ? indices.field_0 : (_temp_var_7 == 1 ? indices.field_1 : (_temp_var_7 == 2 ? indices.field_2 : (_temp_var_7 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_6 == 0 ? indices.field_0 : (_temp_var_6 == 1 ? indices.field_1 : (_temp_var_6 == 2 ? indices.field_2 : (_temp_var_6 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_5 == 0 ? indices.field_0 : (_temp_var_5 == 1 ? indices.field_1 : (_temp_var_5 == 2 ? indices.field_2 : (_temp_var_5 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_17(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_20)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_24;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_24 = _block_k_5_(_env_, _kernel_result_20[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_20[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_20[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_20[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_24 = 37;
    }
        
        _result_[_tid_] = temp_stencil_24;
    }
}



// TODO: There should be a better to check if _block_k_7_ is already defined
#ifndef _block_k_7__func
#define _block_k_7__func
__device__ int _block_k_7_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_8 = ((({ int _temp_var_9 = ((({ int _temp_var_10 = ((values[2] % 4));
        (_temp_var_10 == 0 ? indices.field_0 : (_temp_var_10 == 1 ? indices.field_1 : (_temp_var_10 == 2 ? indices.field_2 : (_temp_var_10 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_9 == 0 ? indices.field_0 : (_temp_var_9 == 1 ? indices.field_1 : (_temp_var_9 == 2 ? indices.field_2 : (_temp_var_9 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_8 == 0 ? indices.field_0 : (_temp_var_8 == 1 ? indices.field_1 : (_temp_var_8 == 2 ? indices.field_2 : (_temp_var_8 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_15(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_18)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_25;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_25 = _block_k_7_(_env_, _kernel_result_18[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_18[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_18[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_18[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_25 = 37;
    }
        
        _result_[_tid_] = temp_stencil_25;
    }
}



// TODO: There should be a better to check if _block_k_9_ is already defined
#ifndef _block_k_9__func
#define _block_k_9__func
__device__ int _block_k_9_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_11 = ((({ int _temp_var_12 = ((({ int _temp_var_13 = ((values[2] % 4));
        (_temp_var_13 == 0 ? indices.field_0 : (_temp_var_13 == 1 ? indices.field_1 : (_temp_var_13 == 2 ? indices.field_2 : (_temp_var_13 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_12 == 0 ? indices.field_0 : (_temp_var_12 == 1 ? indices.field_1 : (_temp_var_12 == 2 ? indices.field_2 : (_temp_var_12 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_11 == 0 ? indices.field_0 : (_temp_var_11 == 1 ? indices.field_1 : (_temp_var_11 == 2 ? indices.field_2 : (_temp_var_11 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_13(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_16)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_26;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_26 = _block_k_9_(_env_, _kernel_result_16[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_16[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_16[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_16[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_26 = 37;
    }
        
        _result_[_tid_] = temp_stencil_26;
    }
}



// TODO: There should be a better to check if _block_k_11_ is already defined
#ifndef _block_k_11__func
#define _block_k_11__func
__device__ int _block_k_11_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_14 = ((({ int _temp_var_15 = ((({ int _temp_var_16 = ((values[2] % 4));
        (_temp_var_16 == 0 ? indices.field_0 : (_temp_var_16 == 1 ? indices.field_1 : (_temp_var_16 == 2 ? indices.field_2 : (_temp_var_16 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_15 == 0 ? indices.field_0 : (_temp_var_15 == 1 ? indices.field_1 : (_temp_var_15 == 2 ? indices.field_2 : (_temp_var_15 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_14 == 0 ? indices.field_0 : (_temp_var_14 == 1 ? indices.field_1 : (_temp_var_14 == 2 ? indices.field_2 : (_temp_var_14 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_11(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_14)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_27;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_27 = _block_k_11_(_env_, _kernel_result_14[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_14[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_14[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_14[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_27 = 37;
    }
        
        _result_[_tid_] = temp_stencil_27;
    }
}



// TODO: There should be a better to check if _block_k_13_ is already defined
#ifndef _block_k_13__func
#define _block_k_13__func
__device__ int _block_k_13_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_17 = ((({ int _temp_var_18 = ((({ int _temp_var_19 = ((values[2] % 4));
        (_temp_var_19 == 0 ? indices.field_0 : (_temp_var_19 == 1 ? indices.field_1 : (_temp_var_19 == 2 ? indices.field_2 : (_temp_var_19 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_18 == 0 ? indices.field_0 : (_temp_var_18 == 1 ? indices.field_1 : (_temp_var_18 == 2 ? indices.field_2 : (_temp_var_18 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_17 == 0 ? indices.field_0 : (_temp_var_17 == 1 ? indices.field_1 : (_temp_var_17 == 2 ? indices.field_2 : (_temp_var_17 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_9(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_12)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_28;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_28 = _block_k_13_(_env_, _kernel_result_12[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_12[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_12[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_12[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_28 = 37;
    }
        
        _result_[_tid_] = temp_stencil_28;
    }
}



// TODO: There should be a better to check if _block_k_15_ is already defined
#ifndef _block_k_15__func
#define _block_k_15__func
__device__ int _block_k_15_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_20 = ((({ int _temp_var_21 = ((({ int _temp_var_22 = ((values[2] % 4));
        (_temp_var_22 == 0 ? indices.field_0 : (_temp_var_22 == 1 ? indices.field_1 : (_temp_var_22 == 2 ? indices.field_2 : (_temp_var_22 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_21 == 0 ? indices.field_0 : (_temp_var_21 == 1 ? indices.field_1 : (_temp_var_21 == 2 ? indices.field_2 : (_temp_var_21 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_20 == 0 ? indices.field_0 : (_temp_var_20 == 1 ? indices.field_1 : (_temp_var_20 == 2 ? indices.field_2 : (_temp_var_20 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_7(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_10)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_29;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_29 = _block_k_15_(_env_, _kernel_result_10[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_10[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_10[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_10[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_29 = 37;
    }
        
        _result_[_tid_] = temp_stencil_29;
    }
}



// TODO: There should be a better to check if _block_k_17_ is already defined
#ifndef _block_k_17__func
#define _block_k_17__func
__device__ int _block_k_17_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_23 = ((({ int _temp_var_24 = ((({ int _temp_var_25 = ((values[2] % 4));
        (_temp_var_25 == 0 ? indices.field_0 : (_temp_var_25 == 1 ? indices.field_1 : (_temp_var_25 == 2 ? indices.field_2 : (_temp_var_25 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_24 == 0 ? indices.field_0 : (_temp_var_24 == 1 ? indices.field_1 : (_temp_var_24 == 2 ? indices.field_2 : (_temp_var_24 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_23 == 0 ? indices.field_0 : (_temp_var_23 == 1 ? indices.field_1 : (_temp_var_23 == 2 ? indices.field_2 : (_temp_var_23 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_5(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_8)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_30;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_30 = _block_k_17_(_env_, _kernel_result_8[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_8[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_8[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_8[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_30 = 37;
    }
        
        _result_[_tid_] = temp_stencil_30;
    }
}



// TODO: There should be a better to check if _block_k_19_ is already defined
#ifndef _block_k_19__func
#define _block_k_19__func
__device__ int _block_k_19_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_26 = ((({ int _temp_var_27 = ((({ int _temp_var_28 = ((values[2] % 4));
        (_temp_var_28 == 0 ? indices.field_0 : (_temp_var_28 == 1 ? indices.field_1 : (_temp_var_28 == 2 ? indices.field_2 : (_temp_var_28 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_27 == 0 ? indices.field_0 : (_temp_var_27 == 1 ? indices.field_1 : (_temp_var_27 == 2 ? indices.field_2 : (_temp_var_27 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_26 == 0 ? indices.field_0 : (_temp_var_26 == 1 ? indices.field_1 : (_temp_var_26 == 2 ? indices.field_2 : (_temp_var_26 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_3(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_6)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_31;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_31 = _block_k_19_(_env_, _kernel_result_6[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_6[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_6[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_6[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_31 = 37;
    }
        
        _result_[_tid_] = temp_stencil_31;
    }
}



// TODO: There should be a better to check if _block_k_21_ is already defined
#ifndef _block_k_21__func
#define _block_k_21__func
__device__ int _block_k_21_(environment_t *_env_, int _values_0, int _values_1, int _values_2, int _values_3, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    // (Re)construct array from separately passed parameters
    int values[] = { _values_0, _values_1, _values_2, _values_3 };
    
    {
        return (((((((values[0] % 938)) + ((values[1] / 97)))) % 97717)) + ((((({ int _temp_var_29 = ((({ int _temp_var_30 = ((({ int _temp_var_31 = ((values[2] % 4));
        (_temp_var_31 == 0 ? indices.field_0 : (_temp_var_31 == 1 ? indices.field_1 : (_temp_var_31 == 2 ? indices.field_2 : (_temp_var_31 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_30 == 0 ? indices.field_0 : (_temp_var_30 == 1 ? indices.field_1 : (_temp_var_30 == 2 ? indices.field_2 : (_temp_var_30 == 3 ? indices.field_3 : NULL)))); }) % 4));
        (_temp_var_29 == 0 ? indices.field_0 : (_temp_var_29 == 1 ? indices.field_1 : (_temp_var_29 == 2 ? indices.field_2 : (_temp_var_29 == 3 ? indices.field_3 : NULL)))); }) * ((values[3] % 7)))) % 99)));
    }
}

#endif


__global__ void kernel_1(environment_t *_env_, int _num_threads_, int *_result_, int *_kernel_result_4)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {
    int temp_stencil_32;

    // Indices for all dimensions
    int temp_stencil_dim_0 = _tid_ / 500000;
int temp_stencil_dim_1 = (_tid_ / 1000) % 500;
int temp_stencil_dim_2 = (_tid_ / 2) % 500;
int temp_stencil_dim_3 = (_tid_ / 1) % 2;

    if (temp_stencil_dim_0 + -1 >= 0 && temp_stencil_dim_0 + 1 < 20 && temp_stencil_dim_1 + -1 >= 0 && temp_stencil_dim_1 + 0 < 500 && temp_stencil_dim_2 + 0 >= 0 && temp_stencil_dim_2 + 0 < 500 && temp_stencil_dim_3 + 0 >= 0 && temp_stencil_dim_3 + 0 < 2)
    {
        // All value indices within bounds
        
        temp_stencil_32 = _block_k_21_(_env_, _kernel_result_4[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + -1) * 500000], _kernel_result_4[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 0) * 500000], _kernel_result_4[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + 0) * 1000 + (temp_stencil_dim_0 + 1) * 500000], _kernel_result_4[(temp_stencil_dim_3 + 0) * 1 + (temp_stencil_dim_2 + 0) * 2 + (temp_stencil_dim_1 + -1) * 1000 + (temp_stencil_dim_0 + -1) * 500000], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
    else
    {
        // At least one index is out of bounds
        temp_stencil_32 = 37;
    }
        
        _result_[_tid_] = temp_stencil_32;
    }
}


#undef checkErrorReturn
#define checkErrorReturn(result_var, expr) \
if (result_var->last_error = expr) \
{\
    hipError_t error = hipGetLastError();\
    printf("!!! Cuda Failure %s:%d (%i): '%s'\n", __FILE__, __LINE__, expr, hipGetErrorString(error));\
    hipDeviceReset();\
    return result_var;\
}

extern "C" EXPORT result_t *launch_kernel(environment_t *host_env)
{
    // CUDA Initialization
    program_result = new result_t();
    program_result->device_allocations = new vector<void*>();

    timeStartMeasure();

    hipError_t cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        program_result->last_error = -1;
        return program_result;
    }

    checkErrorReturn(program_result, hipFree(0));

    timeReportMeasure(program_result, setup_cuda);


    /* Prepare environment */
        /* Allocate device environment and copy over struct */
    environment_t *dev_env;

    timeStartMeasure();
    checkErrorReturn(program_result, hipMalloc(&dev_env, sizeof(environment_t)));
    timeReportMeasure(program_result, allocate_memory);

    timeStartMeasure();
    checkErrorReturn(program_result, hipMemcpy(dev_env, host_env, sizeof(environment_t), hipMemcpyHostToDevice));
    timeReportMeasure(program_result, transfer_memory);
    

    /* Launch all kernels */
        timeStartMeasure();
    int * _kernel_result_22;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_22, (sizeof(int) * 10000000)));
    program_result->device_allocations->push_back(_kernel_result_22);
    timeReportMeasure(program_result, allocate_memory);
    timeStartMeasure();
    kernel_21<<<39063, 256>>>(dev_env, 10000000, _kernel_result_22);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());
    timeReportMeasure(program_result, kernel);    timeStartMeasure();
    int * _kernel_result_20;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_20, (sizeof(int) * 10000000)));
    program_result->device_allocations->push_back(_kernel_result_20);
    timeReportMeasure(program_result, allocate_memory);
    timeStartMeasure();
    kernel_19<<<39063, 256>>>(dev_env, 10000000, _kernel_result_20, _kernel_result_22);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());
    timeReportMeasure(program_result, kernel);    timeStartMeasure();
    int * _kernel_result_18;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_18, (sizeof(int) * 10000000)));
    program_result->device_allocations->push_back(_kernel_result_18);
    timeReportMeasure(program_result, allocate_memory);
    timeStartMeasure();
    kernel_17<<<39063, 256>>>(dev_env, 10000000, _kernel_result_18, _kernel_result_20);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());
    timeReportMeasure(program_result, kernel);    timeStartMeasure();
    int * _kernel_result_16;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_16, (sizeof(int) * 10000000)));
    program_result->device_allocations->push_back(_kernel_result_16);
    timeReportMeasure(program_result, allocate_memory);
    timeStartMeasure();
    kernel_15<<<39063, 256>>>(dev_env, 10000000, _kernel_result_16, _kernel_result_18);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());
    timeReportMeasure(program_result, kernel);    timeStartMeasure();
    int * _kernel_result_14;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_14, (sizeof(int) * 10000000)));
    program_result->device_allocations->push_back(_kernel_result_14);
    timeReportMeasure(program_result, allocate_memory);
    timeStartMeasure();
    kernel_13<<<39063, 256>>>(dev_env, 10000000, _kernel_result_14, _kernel_result_16);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());
    timeReportMeasure(program_result, kernel);    timeStartMeasure();
    int * _kernel_result_12;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_12, (sizeof(int) * 10000000)));
    program_result->device_allocations->push_back(_kernel_result_12);
    timeReportMeasure(program_result, allocate_memory);
    timeStartMeasure();
    kernel_11<<<39063, 256>>>(dev_env, 10000000, _kernel_result_12, _kernel_result_14);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());
    timeReportMeasure(program_result, kernel);    timeStartMeasure();
    int * _kernel_result_10;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_10, (sizeof(int) * 10000000)));
    program_result->device_allocations->push_back(_kernel_result_10);
    timeReportMeasure(program_result, allocate_memory);
    timeStartMeasure();
    kernel_9<<<39063, 256>>>(dev_env, 10000000, _kernel_result_10, _kernel_result_12);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());
    timeReportMeasure(program_result, kernel);    timeStartMeasure();
    int * _kernel_result_8;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_8, (sizeof(int) * 10000000)));
    program_result->device_allocations->push_back(_kernel_result_8);
    timeReportMeasure(program_result, allocate_memory);
    timeStartMeasure();
    kernel_7<<<39063, 256>>>(dev_env, 10000000, _kernel_result_8, _kernel_result_10);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());
    timeReportMeasure(program_result, kernel);    timeStartMeasure();
    int * _kernel_result_6;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_6, (sizeof(int) * 10000000)));
    program_result->device_allocations->push_back(_kernel_result_6);
    timeReportMeasure(program_result, allocate_memory);
    timeStartMeasure();
    kernel_5<<<39063, 256>>>(dev_env, 10000000, _kernel_result_6, _kernel_result_8);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());
    timeReportMeasure(program_result, kernel);    timeStartMeasure();
    int * _kernel_result_4;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_4, (sizeof(int) * 10000000)));
    program_result->device_allocations->push_back(_kernel_result_4);
    timeReportMeasure(program_result, allocate_memory);
    timeStartMeasure();
    kernel_3<<<39063, 256>>>(dev_env, 10000000, _kernel_result_4, _kernel_result_6);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());
    timeReportMeasure(program_result, kernel);    timeStartMeasure();
    int * _kernel_result_2;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_2, (sizeof(int) * 10000000)));
    program_result->device_allocations->push_back(_kernel_result_2);
    timeReportMeasure(program_result, allocate_memory);
    timeStartMeasure();
    kernel_1<<<39063, 256>>>(dev_env, 10000000, _kernel_result_2, _kernel_result_4);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());
    timeReportMeasure(program_result, kernel);

    /* Copy over result to the host */
    program_result->result = ({
    variable_size_array_t device_array = variable_size_array_t((void *) _kernel_result_2, 10000000);
    int * tmp_result = (int *) malloc(sizeof(int) * device_array.size);

    timeStartMeasure();
    checkErrorReturn(program_result, hipMemcpy(tmp_result, device_array.content, sizeof(int) * device_array.size, hipMemcpyDeviceToHost));
    timeReportMeasure(program_result, transfer_memory);

    variable_size_array_t((void *) tmp_result, device_array.size);
});

    /* Free device memory */
        timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_22));
    timeReportMeasure(program_result, free_memory);
    timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_20));
    timeReportMeasure(program_result, free_memory);
    timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_18));
    timeReportMeasure(program_result, free_memory);
    timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_16));
    timeReportMeasure(program_result, free_memory);
    timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_14));
    timeReportMeasure(program_result, free_memory);
    timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_12));
    timeReportMeasure(program_result, free_memory);
    timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_10));
    timeReportMeasure(program_result, free_memory);
    timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_8));
    timeReportMeasure(program_result, free_memory);
    timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_6));
    timeReportMeasure(program_result, free_memory);
    timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_4));
    timeReportMeasure(program_result, free_memory);
    timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_2));
    timeReportMeasure(program_result, free_memory);


    delete program_result->device_allocations;
    
    return program_result;
}
