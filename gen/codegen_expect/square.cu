#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <chrono>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

using namespace std;


/* ----- BEGIN Shared Library Export ----- */
// taken from http://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux

#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT __declspec(dllexport)
    #define IMPORT __declspec(dllimport)
#elif defined(_GCC)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif
/* ----- END Shared Library Export ----- */

/* ----- BEGIN Class Type ----- */
typedef int obj_id_t;
typedef int class_id_t;
/* ----- END Class Type ----- */

/* ----- BEGIN Union Type ----- */
typedef union union_type_value {
    obj_id_t object_id;
    int int_;
    float float_;
    bool bool_;
} union_v_t;

typedef struct union_type_struct
{
    class_id_t class_id;
    union_v_t value;
} union_t;
/* ----- END Union Type ----- */


/* ----- BEGIN Environment (lexical variables) ----- */
// environment_struct must be defined later
typedef struct environment_struct environment_t;
/* ----- END Environment (lexical variables) ----- */


/* ----- BEGIN Forward declarations ----- */
typedef struct result_t result_t;
/* ----- END Forward declarations ----- */


/* ----- BEGIN Macros ----- */
#define checkErrorReturn(result_var, expr) \
if (result_var->last_error = expr) \
{\
    hipError_t error = hipGetLastError();\
    printf("!!! Cuda Failure %s:%d (%i): '%s'\n", __FILE__, __LINE__, expr, hipGetErrorString(error));\
    hipDeviceReset();\
    return result_var;\
}

#define timeStartMeasure() start_time = chrono::high_resolution_clock::now();

#define timeReportMeasure(result_var, variable_name) \
end_time = chrono::high_resolution_clock::now(); \
result_var->time_##variable_name = chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();
/* ----- END Macros ----- */
/* ----- BEGIN Structs ----- */
typedef struct result_t {
    int *result;
    int last_error;

    uint64_t time_setup_cuda;
    uint64_t time_prepare_env;
    uint64_t time_kernel;
    uint64_t time_free_memory;

    // Memory management
    vector<void*> *device_allocations;
} result_t;
/* ----- END Structs ----- */

struct environment_struct
{
    int * b1_base;
    int b1_size;
};
__device__ int _block_k_2_(environment_t *_env_, int value)
{
    
    
    {
        return (value * value);
    }
}


__global__ void kernel_7(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_2_(_env_, _env_->b1_base[_tid_]);
    }
}


extern "C" EXPORT result_t *launch_kernel(environment_t *host_env)
{
    // Variables for measuring time
    chrono::high_resolution_clock::time_point start_time;
    chrono::high_resolution_clock::time_point end_time;

    // CUDA Initialization
    result_t *program_result = (result_t *) malloc(sizeof(result_t));
    program_result->device_allocations = new vector<void*>();

    timeStartMeasure();

    hipError_t cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        program_result->last_error = -1;
        return program_result;
    }

    checkErrorReturn(program_result, hipFree(0));

    timeReportMeasure(program_result, setup_cuda);


    /* Prepare environment */
timeStartMeasure();

    void * temp_ptr_b1_base = host_env->b1_base;
    checkErrorReturn(program_result, hipMalloc((void **) &host_env->b1_base, 40000));
    checkErrorReturn(program_result, hipMemcpy(host_env->b1_base, temp_ptr_b1_base, 40000, hipMemcpyHostToDevice));
    /* Allocate device environment and copy over struct */
    environment_t *dev_env;
    checkErrorReturn(program_result, hipMalloc(&dev_env, sizeof(environment_t)));
    checkErrorReturn(program_result, hipMemcpy(dev_env, host_env, sizeof(environment_t), hipMemcpyHostToDevice));

timeReportMeasure(program_result, prepare_env);

    /* Launch all kernels */
timeStartMeasure();
    int * _kernel_result_8;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_8, (sizeof(int) * 10000)));
    program_result->device_allocations->push_back(_kernel_result_8);
    int * _kernel_result_8_host = (int *) malloc((sizeof(int) * 10000));
    kernel_7<<<40, 256>>>(dev_env, 10000, _kernel_result_8);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());

    checkErrorReturn(program_result, hipMemcpy(_kernel_result_8_host, _kernel_result_8, (sizeof(int) * 10000), hipMemcpyDeviceToHost));

timeReportMeasure(program_result, kernel);

    /* Free device memory */
timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_8));

timeReportMeasure(program_result, free_memory);

    delete program_result->device_allocations;
    
    program_result->result = _kernel_result_8_host;
    return program_result;
}
