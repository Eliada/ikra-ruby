#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

using namespace std;


/* ----- BEGIN Shared Library Export ----- */
// taken from http://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux

#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT __declspec(dllexport)
    #define IMPORT __declspec(dllimport)
#elif defined(_GCC)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif
/* ----- END Shared Library Export ----- */

/* ----- BEGIN Class Type ----- */
typedef int obj_id_t;
typedef int class_id_t;
/* ----- END Class Type ----- */

/* ----- BEGIN Union Type ----- */
typedef struct union_type_struct
{
    class_id_t class_id;
    obj_id_t object_id;
} union_t;
/* ----- END Union Type ----- */


/* ----- BEGIN Environment (lexical variables) ----- */
// environment_struct must be defined later
typedef struct environment_struct environment_t;
/* ----- END Environment (lexical variables) ----- */

struct environment_struct
{
    int * b1j_base;
    int b1j_size;
    int * b1_base;
    int b1_size;
};


__device__ int _block_k_2_(environment_t *_env_, int value)
{
    {
        return (value * value);
    }
}


__global__ void kernel(environment_t *_env_, int *_result_)
{
    _result_[threadIdx.x + blockIdx.x * blockDim.x] = _block_k_2_(_env_, _env_->b1_base[_env_->b1j_base[threadIdx.x + blockIdx.x * blockDim.x]]);
}


extern "C" EXPORT int *launch_kernel(environment_t *host_env)
{
    printf("kernel launched\n");
    
    /* Modify host environment to contain device pointers addresses */
    
    void * temp_ptr_b1j_base = host_env->b1j_base;
    checkCudaErrors(hipMalloc((void **) &host_env->b1j_base, 40000));
    checkCudaErrors(hipMemcpy(host_env->b1j_base, temp_ptr_b1j_base, 40000, hipMemcpyHostToDevice));

    void * temp_ptr_b1_base = host_env->b1_base;
    checkCudaErrors(hipMalloc((void **) &host_env->b1_base, 40000));
    checkCudaErrors(hipMemcpy(host_env->b1_base, temp_ptr_b1_base, 40000, hipMemcpyHostToDevice));


    /* Allocate device environment and copy over struct */
    environment_t *dev_env;
    checkCudaErrors(hipMalloc(&dev_env, sizeof(environment_t)));
    checkCudaErrors(hipMemcpy(dev_env, host_env, sizeof(environment_t), hipMemcpyHostToDevice));

    int *host_result = (int *) malloc(sizeof(int) * 10000);
    int *device_result;
    checkCudaErrors(hipMalloc(&device_result, sizeof(int) * 10000));
    
    dim3 dim_grid(40, 1, 1);
    dim3 dim_block(250, 1, 1);
    
    kernel<<<dim_grid, dim_block>>>(dev_env, device_result);

    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(host_result, device_result, sizeof(int) * 10000, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(dev_env));

    return host_result;
}
