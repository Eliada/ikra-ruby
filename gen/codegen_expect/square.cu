#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

using namespace std;


/* ----- BEGIN Shared Library Export ----- */
// taken from http://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux

#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT __declspec(dllexport)
    #define IMPORT __declspec(dllimport)
#elif defined(_GCC)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif
/* ----- END Shared Library Export ----- */

/* ----- BEGIN Class Type ----- */
typedef int obj_id_t;
typedef int class_id_t;
/* ----- END Class Type ----- */

/* ----- BEGIN Union Type ----- */
typedef union union_type_value {
    obj_id_t object_id;
    int int_;
    float float_;
    bool bool_;
} union_v_t;

typedef struct union_type_struct
{
    class_id_t class_id;
    union_v_t value;
} union_t;
/* ----- END Union Type ----- */


/* ----- BEGIN Environment (lexical variables) ----- */
// environment_struct must be defined later
typedef struct environment_struct environment_t;
/* ----- END Environment (lexical variables) ----- */

struct environment_struct
{
    int * b1_base;
    int b1_size;
};
__device__ int _block_k_2_(environment_t *_env_, int value)
{
    
    {
        return (value * value);
    }
}


__global__ void kernel_7(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_2_(_env_, _env_->b1_base[_tid_]);
    }
}


typedef struct result_t {
    int *result;
    int last_error;
} result_t;

#define checkErrorReturn(result_var, expr) \
if (result_var->last_error = expr) \
{\
    hipError_t error = hipGetLastError();\
    printf("!!! Cuda Failure %s:%d (%i): '%s'\n", __FILE__, __LINE__, expr, hipGetErrorString(error));\
    hipDeviceReset();\
    return result_var;\
}

extern "C" EXPORT result_t *launch_kernel(environment_t *host_env)
{
    // CUDA Initialization
    result_t *program_result = (result_t *) malloc(sizeof(result_t));

    hipError_t cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        program_result->last_error = -1;
        return program_result;
    }

    checkErrorReturn(program_result, hipFree(0));

    /* Prepare environment */

    void * temp_ptr_b1_base = host_env->b1_base;
    checkErrorReturn(program_result, hipMalloc((void **) &host_env->b1_base, 40000));
    checkErrorReturn(program_result, hipMemcpy(host_env->b1_base, temp_ptr_b1_base, 40000, hipMemcpyHostToDevice));
    /* Allocate device environment and copy over struct */
    environment_t *dev_env;
    checkErrorReturn(program_result, hipMalloc(&dev_env, sizeof(environment_t)));
    checkErrorReturn(program_result, hipMemcpy(dev_env, host_env, sizeof(environment_t), hipMemcpyHostToDevice));



    /* Launch all kernels */
    int * _kernel_result_8;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_8, (4 * 10000)));
    int * _kernel_result_8_host = (int *) malloc((4 * 10000));
    kernel_7<<<40, 256>>>(dev_env, 10000, _kernel_result_8);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());

    checkErrorReturn(program_result, hipMemcpy(_kernel_result_8_host, _kernel_result_8, (4 * 10000), hipMemcpyDeviceToHost));


    /* Free device memory */
    checkErrorReturn(program_result, hipFree(_kernel_result_8));

    
    program_result->result = _kernel_result_8_host;
    return program_result;
}
