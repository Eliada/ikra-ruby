#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <chrono>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

using namespace std;


/* ----- BEGIN Shared Library Export ----- */
// taken from http://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux

#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT __declspec(dllexport)
    #define IMPORT __declspec(dllimport)
#elif defined(_GCC)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif
/* ----- END Shared Library Export ----- */

/* ----- BEGIN Class Type ----- */
typedef int obj_id_t;
typedef int class_id_t;
/* ----- END Class Type ----- */

/* ----- BEGIN Union Type ----- */
typedef union union_type_value {
    obj_id_t object_id;
    int int_;
    float float_;
    bool bool_;
} union_v_t;

typedef struct union_type_struct
{
    class_id_t class_id;
    union_v_t value;
} union_t;
/* ----- END Union Type ----- */


/* ----- BEGIN Environment (lexical variables) ----- */
// environment_struct must be defined later
typedef struct environment_struct environment_t;
/* ----- END Environment (lexical variables) ----- */

struct environment_struct
{
    int l2_size;
    int * l2_a;
    int * l2_b;
};
__device__ int _block_k_2_(environment_t *_env_, int index)
{
    
    int i;
    int result;
    int y;
    int x;
    int * lex_b = _env_->l2_b;
    int * lex_a = _env_->l2_a;
    int lex_size = _env_->l2_size;
    {
        x = ((index % lex_size));
        y = ((index / lex_size));
        result = 0;
        for (i = 0; i <= (lex_size - 1); i++)
        {
            result = ((result + ((lex_a[((((y * lex_size)) + i))] * lex_b[((((i * lex_size)) + x))]))));
        }
        i--;
        return result;
    }
}


__global__ void kernel_5(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_2_(_env_, _tid_);
    }
}


typedef struct result_t {
    int *result;
    int last_error;

    uint64_t time_setup_cuda;
    uint64_t time_prepare_env;
    uint64_t time_kernel;
    uint64_t time_free_memory;
} result_t;

#define checkErrorReturn(result_var, expr) \
if (result_var->last_error = expr) \
{\
    hipError_t error = hipGetLastError();\
    printf("!!! Cuda Failure %s:%d (%i): '%s'\n", __FILE__, __LINE__, expr, hipGetErrorString(error));\
    hipDeviceReset();\
    return result_var;\
}

#define timeStartMeasure() start_time = chrono::high_resolution_clock::now();

#define timeReportMeasure(result_var, variable_name) \
end_time = chrono::high_resolution_clock::now(); \
result_var->time_##variable_name = chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

extern "C" EXPORT result_t *launch_kernel(environment_t *host_env)
{
    // Variables for measuring time
    chrono::high_resolution_clock::time_point start_time;
    chrono::high_resolution_clock::time_point end_time;

    // CUDA Initialization
    result_t *program_result = (result_t *) malloc(sizeof(result_t));

    timeStartMeasure();

    hipError_t cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        program_result->last_error = -1;
        return program_result;
    }

    checkErrorReturn(program_result, hipFree(0));

    timeReportMeasure(program_result, setup_cuda);


    /* Prepare environment */
timeStartMeasure();

    void * temp_ptr_l2_a = host_env->l2_a;
    checkErrorReturn(program_result, hipMalloc((void **) &host_env->l2_a, 22500));
    checkErrorReturn(program_result, hipMemcpy(host_env->l2_a, temp_ptr_l2_a, 22500, hipMemcpyHostToDevice));

    void * temp_ptr_l2_b = host_env->l2_b;
    checkErrorReturn(program_result, hipMalloc((void **) &host_env->l2_b, 22500));
    checkErrorReturn(program_result, hipMemcpy(host_env->l2_b, temp_ptr_l2_b, 22500, hipMemcpyHostToDevice));
    /* Allocate device environment and copy over struct */
    environment_t *dev_env;
    checkErrorReturn(program_result, hipMalloc(&dev_env, sizeof(environment_t)));
    checkErrorReturn(program_result, hipMemcpy(dev_env, host_env, sizeof(environment_t), hipMemcpyHostToDevice));

timeReportMeasure(program_result, prepare_env);

    /* Launch all kernels */
timeStartMeasure();
    int * _kernel_result_6;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_6, (sizeof(int) * 5625)));
    int * _kernel_result_6_host = (int *) malloc((sizeof(int) * 5625));
    kernel_5<<<11, 512>>>(dev_env, 5625, _kernel_result_6);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());

    checkErrorReturn(program_result, hipMemcpy(_kernel_result_6_host, _kernel_result_6, (sizeof(int) * 5625), hipMemcpyDeviceToHost));

timeReportMeasure(program_result, kernel);

    /* Free device memory */
timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_6));

timeReportMeasure(program_result, free_memory);

    program_result->result = _kernel_result_6_host;
    return program_result;
}
