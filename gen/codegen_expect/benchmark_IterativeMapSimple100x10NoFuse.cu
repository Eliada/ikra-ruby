#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <chrono>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

using namespace std;


/* ----- BEGIN Shared Library Export ----- */
// taken from http://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux

#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT __declspec(dllexport)
    #define IMPORT __declspec(dllimport)
#elif defined(_GCC)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif
/* ----- END Shared Library Export ----- */

/* ----- BEGIN Class Type ----- */
typedef int obj_id_t;
typedef int class_id_t;
/* ----- END Class Type ----- */

/* ----- BEGIN Environment (lexical variables) ----- */
// environment_struct must be defined later
typedef struct environment_struct environment_t;
/* ----- END Environment (lexical variables) ----- */


/* ----- BEGIN Forward declarations ----- */
typedef struct result_t result_t;
/* ----- END Forward declarations ----- */

// Define program result variable. Also contains benchmark numbers.
result_t *program_result;

// Variables for measuring time
chrono::high_resolution_clock::time_point start_time;
chrono::high_resolution_clock::time_point end_time;

/* ----- BEGIN Macros ----- */
#define timeStartMeasure() start_time = chrono::high_resolution_clock::now();

#define timeReportMeasure(result_var, variable_name) \
end_time = chrono::high_resolution_clock::now(); \
result_var->time_##variable_name = result_var->time_##variable_name + chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();
/* ----- END Macros ----- */
struct indexed_struct_4_lt_int_int_int_int_gt_t
{
    int field_0;
int field_1;
int field_2;
int field_3;
};

/* ----- BEGIN Structs ----- */
struct variable_size_array_t {
    void *content;
    int size;

    variable_size_array_t(void *content_ = NULL, int size_ = 0) : content(content_), size(size_) { }; 

    static const variable_size_array_t error_return_value;
};

// error_return_value is used in case a host section terminates abnormally
const variable_size_array_t variable_size_array_t::error_return_value = 
    variable_size_array_t(NULL, 0);

/* ----- BEGIN Union Type ----- */
typedef union union_type_value {
    obj_id_t object_id;
    int int_;
    float float_;
    bool bool_;
    void *pointer;
    variable_size_array_t variable_size_array;

    __host__ __device__ union_type_value(int value) : int_(value) { };
    __host__ __device__ union_type_value(float value) : float_(value) { };
    __host__ __device__ union_type_value(bool value) : bool_(value) { };
    __host__ __device__ union_type_value(void *value) : pointer(value) { };
    __host__ __device__ union_type_value(variable_size_array_t value) : variable_size_array(value) { };

    __host__ __device__ static union_type_value from_object_id(obj_id_t value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_int(int value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_float(float value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_bool(bool value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_pointer(void *value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_variable_size_array_t(variable_size_array_t value)
    {
        return union_type_value(value);
    }
} union_v_t;

typedef struct union_type_struct
{
    class_id_t class_id;
    union_v_t value;

    __host__ __device__ union_type_struct(
        class_id_t class_id_ = 0, union_v_t value_ = union_v_t(0))
        : class_id(class_id_), value(value_) { };

    static const union_type_struct error_return_value;
} union_t;

// error_return_value is used in case a host section terminates abnormally
const union_type_struct union_t::error_return_value = union_type_struct(0, union_v_t(0));
/* ----- END Union Type ----- */

typedef struct result_t {
    variable_size_array_t result;
    int last_error;

    uint64_t time_setup_cuda;
    uint64_t time_prepare_env;
    uint64_t time_kernel;
    uint64_t time_free_memory;
    uint64_t time_transfer_memory;
    uint64_t time_allocate_memory;

    // Memory management
    vector<void*> *device_allocations;
} result_t;
/* ----- END Structs ----- */

struct array_command_1 {
    // Ikra::Symbolic::ArrayIndexCommand
    indexed_struct_4_lt_int_int_int_int_gt_t *result;
    __host__ __device__ array_command_1(indexed_struct_4_lt_int_int_int_int_gt_t *result = NULL) : result(result) { }
};
struct array_command_2 {
    // Ikra::Symbolic::ArrayCombineCommand
    int *result;
    array_command_1 *input_0;
    __host__ __device__ array_command_2(int *result = NULL, array_command_1 *input_0 = NULL) : result(result), input_0(input_0) { }
};
struct array_command_3 {
    // Ikra::Symbolic::FixedSizeArrayInHostSectionCommand
    int *result;
    variable_size_array_t input_0;
    __host__ __device__ array_command_3(int *result = NULL, variable_size_array_t input_0 = variable_size_array_t::error_return_value) : result(result), input_0(input_0) { }
    int size() { return input_0.size; }
};
struct array_command_5 {
    // Ikra::Symbolic::ArrayIndexCommand
    indexed_struct_4_lt_int_int_int_int_gt_t *result;
    __host__ __device__ array_command_5(indexed_struct_4_lt_int_int_int_int_gt_t *result = NULL) : result(result) { }
};
struct array_command_4 {
    // Ikra::Symbolic::ArrayCombineCommand
    int *result;
    array_command_3 *input_0;
    array_command_5 *input_1;
    __host__ __device__ array_command_4(int *result = NULL, array_command_3 *input_0 = NULL, array_command_5 *input_1 = NULL) : result(result), input_0(input_0), input_1(input_1) { }
};
struct environment_struct
{
};

// TODO: There should be a better to check if _block_k_2_ is already defined
#ifndef _block_k_2__func
#define _block_k_2__func
__device__ int _block_k_2_(environment_t *_env_, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    {
        return (indices.field_2 % 133777);
    }
}

#endif


__global__ void kernel_329(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_2_(_env_, ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_2_ is already defined
#ifndef _block_k_2__func
#define _block_k_2__func
__device__ int _block_k_2_(environment_t *_env_, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    {
        return (indices.field_2 % 133777);
    }
}

#endif


__global__ void kernel_331(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_2_(_env_, ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}




__global__ void kernel_333(environment_t *_env_, int _num_threads_, int *_result_, int *_array_335_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_335_[_tid_];
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_336(environment_t *_env_, int _num_threads_, int *_result_, int *_array_338_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_338_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}




__global__ void kernel_339(environment_t *_env_, int _num_threads_, int *_result_, int *_array_341_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_341_[_tid_];
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_342(environment_t *_env_, int _num_threads_, int *_result_, int *_array_344_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_344_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}




__global__ void kernel_345(environment_t *_env_, int _num_threads_, int *_result_, int *_array_347_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_347_[_tid_];
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_348(environment_t *_env_, int _num_threads_, int *_result_, int *_array_350_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_350_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}




__global__ void kernel_351(environment_t *_env_, int _num_threads_, int *_result_, int *_array_353_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_353_[_tid_];
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_354(environment_t *_env_, int _num_threads_, int *_result_, int *_array_356_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_356_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_357(environment_t *_env_, int _num_threads_, int *_result_, int *_array_359_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_359_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_360(environment_t *_env_, int _num_threads_, int *_result_, int *_array_362_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_362_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_363(environment_t *_env_, int _num_threads_, int *_result_, int *_array_365_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_365_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_366(environment_t *_env_, int _num_threads_, int *_result_, int *_array_368_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_368_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_369(environment_t *_env_, int _num_threads_, int *_result_, int *_array_371_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_371_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_372(environment_t *_env_, int _num_threads_, int *_result_, int *_array_374_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_374_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_375(environment_t *_env_, int _num_threads_, int *_result_, int *_array_377_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_377_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_378(environment_t *_env_, int _num_threads_, int *_result_, int *_array_380_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_380_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_381(environment_t *_env_, int _num_threads_, int *_result_, int *_array_383_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_383_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_384(environment_t *_env_, int _num_threads_, int *_result_, int *_array_386_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_386_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_387(environment_t *_env_, int _num_threads_, int *_result_, int *_array_389_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_389_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_390(environment_t *_env_, int _num_threads_, int *_result_, int *_array_392_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_392_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_393(environment_t *_env_, int _num_threads_, int *_result_, int *_array_395_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_395_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_396(environment_t *_env_, int _num_threads_, int *_result_, int *_array_398_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_398_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_399(environment_t *_env_, int _num_threads_, int *_result_, int *_array_401_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_401_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_402(environment_t *_env_, int _num_threads_, int *_result_, int *_array_404_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_404_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}




__global__ void kernel_405(environment_t *_env_, int _num_threads_, int *_result_, int *_array_407_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _array_407_[_tid_];
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((i + indices.field_2)) % 13377);
    }
}

#endif


__global__ void kernel_408(environment_t *_env_, int _num_threads_, int *_result_, int *_array_410_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_410_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}


#undef checkErrorReturn
#define checkErrorReturn(result_var, expr) \
if (result_var->last_error = expr) \
{\
    hipError_t error = hipGetLastError();\
    printf("!!! Cuda Failure %s:%d (%i): '%s'\n", __FILE__, __LINE__, expr, hipGetErrorString(error));\
    hipDeviceReset();\
    return variable_size_array_t::error_return_value;\
}

variable_size_array_t _host_section__(environment_t *host_env, environment_t *dev_env, result_t *program_result)
{
    array_command_2 * x = new array_command_2();
    int r;
    array_command_4 * _ssa_var_old_data_12;
    array_command_4 * _ssa_var_y_11;
    array_command_4 * _ssa_var_old_data_10;
    array_command_4 * _ssa_var_y_9;
    array_command_4 * _ssa_var_old_data_8;
    array_command_4 * _ssa_var_y_7;
    array_command_4 * _ssa_var_old_data_6;
    array_command_4 * _ssa_var_y_5;
    array_command_4 * _ssa_var_old_data_4;
    array_command_4 * _ssa_var_y_3;
    union_t _ssa_var_old_data_2;
    union_t _ssa_var_y_1;
    {
        _ssa_var_y_1 = union_t(10, union_v_t::from_pointer((void *) new array_command_3(NULL, ({
            // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]
        
            array_command_2 * cmd = x;
        
            if (cmd->result == 0) {
                    timeStartMeasure();
            int * _kernel_result_330;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_330, (sizeof(int) * 10000000)));
            program_result->device_allocations->push_back(_kernel_result_330);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_329<<<39063, 256>>>(dev_env, 10000000, _kernel_result_330);
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);
                cmd->result = _kernel_result_330;
        
                
            }
        
            variable_size_array_t((void *) cmd->result, 10000000);
        }))));
        _ssa_var_old_data_2 = union_t(19, union_v_t::from_pointer((void *) x));
        for (r = 0; r <= (100 - 1); r++)
        {
            _ssa_var_y_3 = new array_command_4(NULL, new array_command_3(NULL, ({
                variable_size_array_t _polytemp_result_49;
                {
                    union_t _polytemp_expr_50 = _ssa_var_y_1;
                    switch (_polytemp_expr_50.class_id)
                    {
                        case 10: /* [Ikra::Symbolic::FixedSizeArrayInHostSectionCommand, size = 10000000] (Ikra::Symbolic::ArrayCommand) */ _polytemp_result_49 = ({
                            // [Ikra::Symbolic::FixedSizeArrayInHostSectionCommand, size = 10000000]
                        
                            array_command_3 * cmd = (array_command_3 *) _polytemp_expr_50.value.pointer;
                        
                            if (cmd->result == 0) {
                                    timeStartMeasure();
                            int * _kernel_result_334;
                            checkErrorReturn(program_result, hipMalloc(&_kernel_result_334, (sizeof(int) * 10000000)));
                            program_result->device_allocations->push_back(_kernel_result_334);
                            timeReportMeasure(program_result, allocate_memory);
                            timeStartMeasure();
                            kernel_333<<<39063, 256>>>(dev_env, 10000000, _kernel_result_334, ((int *) cmd->input_0.content));
                            checkErrorReturn(program_result, hipPeekAtLastError());
                            checkErrorReturn(program_result, hipDeviceSynchronize());
                            timeReportMeasure(program_result, kernel);
                                cmd->result = _kernel_result_334;
                        
                                
                            }
                        
                            variable_size_array_t((void *) cmd->result, 10000000);
                        }); break;
                        case 20: /* [Ikra::Symbolic::ArrayCombineCommand, size = 10000000] (Ikra::Symbolic::ArrayCommand) */ _polytemp_result_49 = ({
                            // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [SendNode: [SendNode: [LVarReadNode: _ssa_var_y_1].__call__()].to_command()].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
                        
                            array_command_4 * cmd = (array_command_4 *) _polytemp_expr_50.value.pointer;
                        
                            if (cmd->result == 0) {
                                    timeStartMeasure();
                            int * _kernel_result_337;
                            checkErrorReturn(program_result, hipMalloc(&_kernel_result_337, (sizeof(int) * 10000000)));
                            program_result->device_allocations->push_back(_kernel_result_337);
                            timeReportMeasure(program_result, allocate_memory);
                            timeStartMeasure();
                            kernel_336<<<39063, 256>>>(dev_env, 10000000, _kernel_result_337, ((int *) ((int *) cmd->input_0->input_0.content)));
                            checkErrorReturn(program_result, hipPeekAtLastError());
                            checkErrorReturn(program_result, hipDeviceSynchronize());
                            timeReportMeasure(program_result, kernel);
                                cmd->result = _kernel_result_337;
                        
                                
                            }
                        
                            variable_size_array_t((void *) cmd->result, 10000000);
                        }); break;
                    }
                }
                _polytemp_result_49;
            })));
            ({
                bool _polytemp_result_57;
                {
                    union_t _polytemp_expr_58 = _ssa_var_old_data_2;
                    switch (_polytemp_expr_58.class_id)
                    {
                        case 19: /* [Ikra::Symbolic::ArrayCombineCommand, size = 10000000] (Ikra::Symbolic::ArrayCommand) */ _polytemp_result_57 = ({
                            array_command_2 * cmd_to_free = (array_command_2 *) _polytemp_expr_58.value.pointer;
                        
                            timeStartMeasure();
                            bool freed_memory = false;
                        
                            if (cmd_to_free->result != 0) {
                                checkErrorReturn(program_result, hipFree(cmd_to_free->result));;
                        
                                // Remove from list of allocations
                                program_result->device_allocations->erase(
                                    std::remove(
                                        program_result->device_allocations->begin(),
                                        program_result->device_allocations->end(),
                                        cmd_to_free->result),
                                    program_result->device_allocations->end());
                        
                                freed_memory = true;
                            }
                        
                            timeReportMeasure(program_result, free_memory);
                            
                            freed_memory;
                        }); break;
                        case 20: /* [Ikra::Symbolic::ArrayCombineCommand, size = 10000000] (Ikra::Symbolic::ArrayCommand) */ _polytemp_result_57 = ({
                            array_command_4 * cmd_to_free = (array_command_4 *) _polytemp_expr_58.value.pointer;
                        
                            timeStartMeasure();
                            bool freed_memory = false;
                        
                            if (cmd_to_free->result != 0) {
                                checkErrorReturn(program_result, hipFree(cmd_to_free->result));;
                        
                                // Remove from list of allocations
                                program_result->device_allocations->erase(
                                    std::remove(
                                        program_result->device_allocations->begin(),
                                        program_result->device_allocations->end(),
                                        cmd_to_free->result),
                                    program_result->device_allocations->end());
                        
                                freed_memory = true;
                            }
                        
                            timeReportMeasure(program_result, free_memory);
                            
                            freed_memory;
                        }); break;
                    }
                }
                _polytemp_result_57;
            });
            _ssa_var_old_data_4 = _ssa_var_y_3;
            _ssa_var_y_5 = new array_command_4(NULL, new array_command_3(NULL, ({
                // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [SendNode: [SendNode: [LVarReadNode: _ssa_var_y_1].__call__()].to_command()].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
            
                array_command_4 * cmd = _ssa_var_y_3;
            
                if (cmd->result == 0) {
                        timeStartMeasure();
                int * _kernel_result_358;
                checkErrorReturn(program_result, hipMalloc(&_kernel_result_358, (sizeof(int) * 10000000)));
                program_result->device_allocations->push_back(_kernel_result_358);
                timeReportMeasure(program_result, allocate_memory);
                timeStartMeasure();
                kernel_357<<<39063, 256>>>(dev_env, 10000000, _kernel_result_358, ((int *) ((int *) cmd->input_0->input_0.content)));
                checkErrorReturn(program_result, hipPeekAtLastError());
                checkErrorReturn(program_result, hipDeviceSynchronize());
                timeReportMeasure(program_result, kernel);
                    cmd->result = _kernel_result_358;
            
                    
                }
            
                variable_size_array_t((void *) cmd->result, 10000000);
            })));
            ({
                array_command_4 * cmd_to_free = _ssa_var_old_data_4;
            
                timeStartMeasure();
                bool freed_memory = false;
            
                if (cmd_to_free->result != 0) {
                    checkErrorReturn(program_result, hipFree(cmd_to_free->result));;
            
                    // Remove from list of allocations
                    program_result->device_allocations->erase(
                        std::remove(
                            program_result->device_allocations->begin(),
                            program_result->device_allocations->end(),
                            cmd_to_free->result),
                        program_result->device_allocations->end());
            
                    freed_memory = true;
                }
            
                timeReportMeasure(program_result, free_memory);
                
                freed_memory;
            });
            _ssa_var_old_data_6 = _ssa_var_y_5;
            _ssa_var_y_7 = new array_command_4(NULL, new array_command_3(NULL, ({
                // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [SendNode: [SendNode: [LVarReadNode: _ssa_var_y_1].__call__()].to_command()].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
            
                array_command_4 * cmd = _ssa_var_y_5;
            
                if (cmd->result == 0) {
                        timeStartMeasure();
                int * _kernel_result_370;
                checkErrorReturn(program_result, hipMalloc(&_kernel_result_370, (sizeof(int) * 10000000)));
                program_result->device_allocations->push_back(_kernel_result_370);
                timeReportMeasure(program_result, allocate_memory);
                timeStartMeasure();
                kernel_369<<<39063, 256>>>(dev_env, 10000000, _kernel_result_370, ((int *) ((int *) cmd->input_0->input_0.content)));
                checkErrorReturn(program_result, hipPeekAtLastError());
                checkErrorReturn(program_result, hipDeviceSynchronize());
                timeReportMeasure(program_result, kernel);
                    cmd->result = _kernel_result_370;
            
                    
                }
            
                variable_size_array_t((void *) cmd->result, 10000000);
            })));
            ({
                array_command_4 * cmd_to_free = _ssa_var_old_data_6;
            
                timeStartMeasure();
                bool freed_memory = false;
            
                if (cmd_to_free->result != 0) {
                    checkErrorReturn(program_result, hipFree(cmd_to_free->result));;
            
                    // Remove from list of allocations
                    program_result->device_allocations->erase(
                        std::remove(
                            program_result->device_allocations->begin(),
                            program_result->device_allocations->end(),
                            cmd_to_free->result),
                        program_result->device_allocations->end());
            
                    freed_memory = true;
                }
            
                timeReportMeasure(program_result, free_memory);
                
                freed_memory;
            });
            _ssa_var_old_data_8 = _ssa_var_y_7;
            _ssa_var_y_9 = new array_command_4(NULL, new array_command_3(NULL, ({
                // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [SendNode: [SendNode: [LVarReadNode: _ssa_var_y_1].__call__()].to_command()].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
            
                array_command_4 * cmd = _ssa_var_y_7;
            
                if (cmd->result == 0) {
                        timeStartMeasure();
                int * _kernel_result_382;
                checkErrorReturn(program_result, hipMalloc(&_kernel_result_382, (sizeof(int) * 10000000)));
                program_result->device_allocations->push_back(_kernel_result_382);
                timeReportMeasure(program_result, allocate_memory);
                timeStartMeasure();
                kernel_381<<<39063, 256>>>(dev_env, 10000000, _kernel_result_382, ((int *) ((int *) cmd->input_0->input_0.content)));
                checkErrorReturn(program_result, hipPeekAtLastError());
                checkErrorReturn(program_result, hipDeviceSynchronize());
                timeReportMeasure(program_result, kernel);
                    cmd->result = _kernel_result_382;
            
                    
                }
            
                variable_size_array_t((void *) cmd->result, 10000000);
            })));
            ({
                array_command_4 * cmd_to_free = _ssa_var_old_data_8;
            
                timeStartMeasure();
                bool freed_memory = false;
            
                if (cmd_to_free->result != 0) {
                    checkErrorReturn(program_result, hipFree(cmd_to_free->result));;
            
                    // Remove from list of allocations
                    program_result->device_allocations->erase(
                        std::remove(
                            program_result->device_allocations->begin(),
                            program_result->device_allocations->end(),
                            cmd_to_free->result),
                        program_result->device_allocations->end());
            
                    freed_memory = true;
                }
            
                timeReportMeasure(program_result, free_memory);
                
                freed_memory;
            });
            _ssa_var_old_data_10 = _ssa_var_y_9;
            _ssa_var_y_11 = new array_command_4(NULL, new array_command_3(NULL, ({
                // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [SendNode: [SendNode: [LVarReadNode: _ssa_var_y_1].__call__()].to_command()].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
            
                array_command_4 * cmd = _ssa_var_y_9;
            
                if (cmd->result == 0) {
                        timeStartMeasure();
                int * _kernel_result_394;
                checkErrorReturn(program_result, hipMalloc(&_kernel_result_394, (sizeof(int) * 10000000)));
                program_result->device_allocations->push_back(_kernel_result_394);
                timeReportMeasure(program_result, allocate_memory);
                timeStartMeasure();
                kernel_393<<<39063, 256>>>(dev_env, 10000000, _kernel_result_394, ((int *) ((int *) cmd->input_0->input_0.content)));
                checkErrorReturn(program_result, hipPeekAtLastError());
                checkErrorReturn(program_result, hipDeviceSynchronize());
                timeReportMeasure(program_result, kernel);
                    cmd->result = _kernel_result_394;
            
                    
                }
            
                variable_size_array_t((void *) cmd->result, 10000000);
            })));
            ({
                array_command_4 * cmd_to_free = _ssa_var_old_data_10;
            
                timeStartMeasure();
                bool freed_memory = false;
            
                if (cmd_to_free->result != 0) {
                    checkErrorReturn(program_result, hipFree(cmd_to_free->result));;
            
                    // Remove from list of allocations
                    program_result->device_allocations->erase(
                        std::remove(
                            program_result->device_allocations->begin(),
                            program_result->device_allocations->end(),
                            cmd_to_free->result),
                        program_result->device_allocations->end());
            
                    freed_memory = true;
                }
            
                timeReportMeasure(program_result, free_memory);
                
                freed_memory;
            });
            _ssa_var_old_data_12 = _ssa_var_y_11;
            _ssa_var_y_1 = union_t(20, union_v_t::from_pointer((void *) _ssa_var_y_11));
            _ssa_var_old_data_2 = union_t(20, union_v_t::from_pointer((void *) _ssa_var_old_data_12));
        }
        r--;
        return ({
            variable_size_array_t _polytemp_result_59;
            {
                union_t _polytemp_expr_60 = _ssa_var_y_1;
                switch (_polytemp_expr_60.class_id)
                {
                    case 10: /* [Ikra::Symbolic::FixedSizeArrayInHostSectionCommand, size = 10000000] (Ikra::Symbolic::ArrayCommand) */ _polytemp_result_59 = ({
                        // [Ikra::Symbolic::FixedSizeArrayInHostSectionCommand, size = 10000000]
                    
                        array_command_3 * cmd = (array_command_3 *) _polytemp_expr_60.value.pointer;
                    
                        if (cmd->result == 0) {
                                timeStartMeasure();
                        int * _kernel_result_406;
                        checkErrorReturn(program_result, hipMalloc(&_kernel_result_406, (sizeof(int) * 10000000)));
                        program_result->device_allocations->push_back(_kernel_result_406);
                        timeReportMeasure(program_result, allocate_memory);
                        timeStartMeasure();
                        kernel_405<<<39063, 256>>>(dev_env, 10000000, _kernel_result_406, ((int *) cmd->input_0.content));
                        checkErrorReturn(program_result, hipPeekAtLastError());
                        checkErrorReturn(program_result, hipDeviceSynchronize());
                        timeReportMeasure(program_result, kernel);
                            cmd->result = _kernel_result_406;
                    
                            
                        }
                    
                        variable_size_array_t((void *) cmd->result, 10000000);
                    }); break;
                    case 20: /* [Ikra::Symbolic::ArrayCombineCommand, size = 10000000] (Ikra::Symbolic::ArrayCommand) */ _polytemp_result_59 = ({
                        // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [SendNode: [SendNode: [LVarReadNode: _ssa_var_y_1].__call__()].to_command()].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
                    
                        array_command_4 * cmd = (array_command_4 *) _polytemp_expr_60.value.pointer;
                    
                        if (cmd->result == 0) {
                                timeStartMeasure();
                        int * _kernel_result_409;
                        checkErrorReturn(program_result, hipMalloc(&_kernel_result_409, (sizeof(int) * 10000000)));
                        program_result->device_allocations->push_back(_kernel_result_409);
                        timeReportMeasure(program_result, allocate_memory);
                        timeStartMeasure();
                        kernel_408<<<39063, 256>>>(dev_env, 10000000, _kernel_result_409, ((int *) ((int *) cmd->input_0->input_0.content)));
                        checkErrorReturn(program_result, hipPeekAtLastError());
                        checkErrorReturn(program_result, hipDeviceSynchronize());
                        timeReportMeasure(program_result, kernel);
                            cmd->result = _kernel_result_409;
                    
                            
                        }
                    
                        variable_size_array_t((void *) cmd->result, 10000000);
                    }); break;
                }
            }
            _polytemp_result_59;
        });
    }
}

#undef checkErrorReturn
#define checkErrorReturn(result_var, expr) \
expr

extern "C" EXPORT result_t *launch_kernel(environment_t *host_env)
{
    // CUDA Initialization
    program_result = new result_t();
    program_result->device_allocations = new vector<void*>();

    timeStartMeasure();

    hipError_t cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        program_result->last_error = -1;
        return program_result;
    }

    checkErrorReturn(program_result, hipFree(0));

    timeReportMeasure(program_result, setup_cuda);


    /* Prepare environment */
        /* Allocate device environment and copy over struct */
    environment_t *dev_env;

    timeStartMeasure();
    checkErrorReturn(program_result, hipMalloc(&dev_env, sizeof(environment_t)));
    timeReportMeasure(program_result, allocate_memory);

    timeStartMeasure();
    checkErrorReturn(program_result, hipMemcpy(dev_env, host_env, sizeof(environment_t), hipMemcpyHostToDevice));
    timeReportMeasure(program_result, transfer_memory);
    


    /* Copy back memory and set pointer of result */
    program_result->result = ({
    variable_size_array_t device_array = _host_section__(host_env, dev_env, program_result);
    int * tmp_result = (int *) malloc(sizeof(int) * device_array.size);

    timeStartMeasure();
    checkErrorReturn(program_result, hipMemcpy(tmp_result, device_array.content, sizeof(int) * device_array.size, hipMemcpyDeviceToHost));
    timeReportMeasure(program_result, transfer_memory);

    variable_size_array_t((void *) tmp_result, device_array.size);
});

    /* Free device memory */
    timeStartMeasure();

    for (
        auto device_ptr = program_result->device_allocations->begin(); 
        device_ptr < program_result->device_allocations->end(); 
        device_ptr++)
    {
        checkErrorReturn(program_result, hipFree(*device_ptr));
    }

    delete program_result->device_allocations;

    timeReportMeasure(program_result, free_memory);

    return program_result;
}
