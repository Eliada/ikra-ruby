#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

using namespace std;


/* ----- BEGIN Shared Library Export ----- */
// taken from http://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux

#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT __declspec(dllexport)
    #define IMPORT __declspec(dllimport)
#elif defined(_GCC)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif
/* ----- END Shared Library Export ----- */

/* ----- BEGIN Class Type ----- */
typedef int obj_id_t;
typedef int class_id_t;
/* ----- END Class Type ----- */

/* ----- BEGIN Union Type ----- */
typedef union union_type_value {
    obj_id_t object_id;
    int int_;
    float float_;
    bool bool_;
} union_v_t;

typedef struct union_type_struct
{
    class_id_t class_id;
    union_v_t value;
} union_t;
/* ----- END Union Type ----- */


/* ----- BEGIN Environment (lexical variables) ----- */
// environment_struct must be defined later
typedef struct environment_struct environment_t;
/* ----- END Environment (lexical variables) ----- */

struct environment_struct
{
    int l1_size;
    int * l1_a;
    int * l1_b;
};
__device__ int _block_k_1_(environment_t *_env_, int index)
{
    
    int i;
    int result;
    int y;
    int x;
    int * lex_b = _env_->l1_b;
    int * lex_a = _env_->l1_a;
    int lex_size = _env_->l1_size;
    {
        x = ((index % lex_size));
        y = ((index / lex_size));
        result = 0;
        for (i = 0; i <= (lex_size - 1); i++)
        {
            result = ((result + ((lex_a[((((y * lex_size)) + i))] * lex_b[((((i * lex_size)) + x))]))));
        }
        i--;
        return result;
    }
}


__global__ void kernel_3(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_1_(_env_, _tid_);
    }
}


typedef struct result_t {
    int *result;
    int last_error;
} result_t;

#define checkErrorReturn(result_var, expr) \
if (result_var->last_error = expr) \
{\
    hipError_t error = hipGetLastError();\
    printf("!!! Cuda Failure %s:%d (%i): '%s'\n", __FILE__, __LINE__, expr, hipGetErrorString(error));\
    hipDeviceReset();\
    return result_var;\
}

extern "C" EXPORT result_t *launch_kernel(environment_t *host_env)
{
    // CUDA Initialization
    result_t *program_result = (result_t *) malloc(sizeof(result_t));

    hipError_t cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        program_result->last_error = -1;
        return program_result;
    }

    checkErrorReturn(program_result, hipFree(0));

    /* Prepare environment */

    void * temp_ptr_l1_a = host_env->l1_a;
    checkErrorReturn(program_result, hipMalloc((void **) &host_env->l1_a, 400));
    checkErrorReturn(program_result, hipMemcpy(host_env->l1_a, temp_ptr_l1_a, 400, hipMemcpyHostToDevice));

    void * temp_ptr_l1_b = host_env->l1_b;
    checkErrorReturn(program_result, hipMalloc((void **) &host_env->l1_b, 400));
    checkErrorReturn(program_result, hipMemcpy(host_env->l1_b, temp_ptr_l1_b, 400, hipMemcpyHostToDevice));
    /* Allocate device environment and copy over struct */
    environment_t *dev_env;
    checkErrorReturn(program_result, hipMalloc(&dev_env, sizeof(environment_t)));
    checkErrorReturn(program_result, hipMemcpy(dev_env, host_env, sizeof(environment_t), hipMemcpyHostToDevice));



    /* Launch all kernels */
    int * _kernel_result_4;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_4, (4 * 100)));
    int * _kernel_result_4_host = (int *) malloc((4 * 100));
    kernel_3<<<1, 100>>>(dev_env, 100, _kernel_result_4);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());

    checkErrorReturn(program_result, hipMemcpy(_kernel_result_4_host, _kernel_result_4, (4 * 100), hipMemcpyDeviceToHost));


    /* Free device memory */
    checkErrorReturn(program_result, hipFree(_kernel_result_4));

    
    program_result->result = _kernel_result_4_host;
    return program_result;
}
