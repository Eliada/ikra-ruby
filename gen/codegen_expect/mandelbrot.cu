#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <chrono>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

using namespace std;


/* ----- BEGIN Shared Library Export ----- */
// taken from http://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux

#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT __declspec(dllexport)
    #define IMPORT __declspec(dllimport)
#elif defined(_GCC)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif
/* ----- END Shared Library Export ----- */

/* ----- BEGIN Class Type ----- */
typedef int obj_id_t;
typedef int class_id_t;
/* ----- END Class Type ----- */

/* ----- BEGIN Environment (lexical variables) ----- */
// environment_struct must be defined later
typedef struct environment_struct environment_t;
/* ----- END Environment (lexical variables) ----- */


/* ----- BEGIN Forward declarations ----- */
typedef struct result_t result_t;
/* ----- END Forward declarations ----- */


/* ----- BEGIN Macros ----- */
#define timeStartMeasure() start_time = chrono::high_resolution_clock::now();

#define timeReportMeasure(result_var, variable_name) \
end_time = chrono::high_resolution_clock::now(); \
result_var->time_##variable_name = chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();
/* ----- END Macros ----- */
/* ----- BEGIN Structs ----- */
template <typename T>
struct array_command_t {
    T *result;
};

struct fixed_size_array_t {
    void *content;
    int size;

    fixed_size_array_t(void *content_ = NULL, int size_ = 0) : content(content_), size(size_) { }; 

    static const fixed_size_array_t error_return_value;
};

// error_return_value is used in case a host section terminates abnormally
const fixed_size_array_t fixed_size_array_t::error_return_value = 
    fixed_size_array_t(NULL, 0);

/* ----- BEGIN Union Type ----- */
typedef union union_type_value {
    obj_id_t object_id;
    int int_;
    float float_;
    bool bool_;
    array_command_t<void> *array_command;
    fixed_size_array_t fixed_size_array;

    __host__ __device__ union_type_value(int value) : int_(value) { };
    __host__ __device__ union_type_value(float value) : float_(value) { };
    __host__ __device__ union_type_value(bool value) : bool_(value) { };
    __host__ __device__ union_type_value(array_command_t<void> *value) : array_command(value) { };
    __host__ __device__ union_type_value(fixed_size_array_t value) : fixed_size_array(value) { };

    __host__ __device__ static union_type_value from_object_id(obj_id_t value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_int(int value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_float(float value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_bool(bool value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_array_command_t(array_command_t<void> *value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_fixed_size_array_t(fixed_size_array_t value)
    {
        return union_type_value(value);
    }
} union_v_t;

typedef struct union_type_struct
{
    class_id_t class_id;
    union_v_t value;

    __host__ __device__ union_type_struct(
        class_id_t class_id_ = 0, union_v_t value_ = union_v_t(0))
        : class_id(class_id_), value(value_) { };

    static const union_type_struct error_return_value;
} union_t;

// error_return_value is used in case a host section terminates abnormally
const union_type_struct union_t::error_return_value = union_type_struct(0, union_v_t(0));
/* ----- END Union Type ----- */

typedef struct result_t {
    fixed_size_array_t result;
    int last_error;

    uint64_t time_setup_cuda;
    uint64_t time_prepare_env;
    uint64_t time_kernel;
    uint64_t time_free_memory;

    // Memory management
    vector<void*> *device_allocations;
} result_t;
/* ----- END Structs ----- */

struct environment_struct
{
    int l2_hx_res;
    float l2_magnify;
    int l2_hy_res;
    int l2_iter_max;
    int l3_inverted;
};



// TODO: There should be a better to check if _block_k_2_ is already defined
#ifndef _block_k_2__func
#define _block_k_2__func
__device__ int _block_k_2_(environment_t *_env_, int j)
{
    
    
    float xx;
    int iter;
    float y;
    float x;
    float cy;
    float cx;
    int hy;
    int hx;
    int lex_iter_max = _env_->l2_iter_max;
    int lex_hy_res = _env_->l2_hy_res;
    float lex_magnify = _env_->l2_magnify;
    int lex_hx_res = _env_->l2_hx_res;
    {
        hx = ((j % lex_hx_res));
        hy = ((j / lex_hx_res));
        cx = ((((((((((((float) hx) / ((float) lex_hx_res))) - 0.5)) / lex_magnify)) * 3.0)) - 0.7));
        cy = ((((((((((float) hy) / ((float) lex_hy_res))) - 0.5)) / lex_magnify)) * 3.0));
        x = 0.0;
        y = 0.0;
        for (iter = 0; iter <= lex_iter_max; iter++)
        {
            {
                xx = ((((((x * x)) - ((y * y)))) + cx));
                y = ((((((2.0 * x)) * y)) + cy));
                x = xx;
                if (((((((x * x)) + ((y * y)))) > 100)))
                {
                
                }
            }
        }
        iter--;
        {
            return (iter % 256);
        }
    }
}

#endif



// TODO: There should be a better to check if _block_k_3_ is already defined
#ifndef _block_k_3__func
#define _block_k_3__func
__device__ int _block_k_3_(environment_t *_env_, int color)
{
    
    
    int lex_inverted = _env_->l3_inverted;
    if (((lex_inverted == 1)))
    {
        {
            {
                return (255 - color);
            }
        }
    }
    else
    {
        {
            return color;
        }
    }
}

#endif


__global__ void kernel_1(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_3_(_env_, _block_k_2_(_env_, _tid_));
    }
}


#undef checkErrorReturn
#define checkErrorReturn(result_var, expr) \
if (result_var->last_error = expr) \
{\
    hipError_t error = hipGetLastError();\
    printf("!!! Cuda Failure %s:%d (%i): '%s'\n", __FILE__, __LINE__, expr, hipGetErrorString(error));\
    hipDeviceReset();\
    return result_var;\
}

extern "C" EXPORT result_t *launch_kernel(environment_t *host_env)
{
    // Variables for measuring time
    chrono::high_resolution_clock::time_point start_time;
    chrono::high_resolution_clock::time_point end_time;

    // CUDA Initialization
    result_t *program_result = (result_t *) malloc(sizeof(result_t));
    program_result->device_allocations = new vector<void*>();

    timeStartMeasure();

    hipError_t cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        program_result->last_error = -1;
        return program_result;
    }

    checkErrorReturn(program_result, hipFree(0));

    timeReportMeasure(program_result, setup_cuda);


    /* Prepare environment */
    timeStartMeasure();
        /* Allocate device environment and copy over struct */
    environment_t *dev_env;
    checkErrorReturn(program_result, hipMalloc(&dev_env, sizeof(environment_t)));
    checkErrorReturn(program_result, hipMemcpy(dev_env, host_env, sizeof(environment_t), hipMemcpyHostToDevice));

    timeReportMeasure(program_result, prepare_env);

    /* Launch all kernels */
    timeStartMeasure();
        int * _kernel_result_2;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_2, (sizeof(int) * 40000)));
    program_result->device_allocations->push_back(_kernel_result_2);
    kernel_1<<<157, 256>>>(dev_env, 40000, _kernel_result_2);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());


    timeReportMeasure(program_result, kernel);

    /* Copy over result to the host */
    program_result->result = ({
    fixed_size_array_t device_array = fixed_size_array_t((void *) _kernel_result_2, 40000);
    int * tmp_result = (int *) malloc(sizeof(int) * device_array.size);
    checkErrorReturn(program_result, hipMemcpy(tmp_result, device_array.content, sizeof(int) * device_array.size, hipMemcpyDeviceToHost));
    fixed_size_array_t((void *) tmp_result, device_array.size);
});

    /* Free device memory */
    timeStartMeasure();
        checkErrorReturn(program_result, hipFree(_kernel_result_2));

    timeReportMeasure(program_result, free_memory);

    delete program_result->device_allocations;
    
    return program_result;
}
