#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

using namespace std;


/* ----- BEGIN Shared Library Export ----- */
// taken from http://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux

#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT __declspec(dllexport)
    #define IMPORT __declspec(dllimport)
#elif defined(_GCC)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif
/* ----- END Shared Library Export ----- */

/* ----- BEGIN Class Type ----- */
typedef int obj_id_t;
typedef int class_id_t;
/* ----- END Class Type ----- */

/* ----- BEGIN Union Type ----- */
typedef union union_type_value {
    obj_id_t object_id;
    int int_;
    float float_;
    bool bool_;
} union_v_t;

typedef struct union_type_struct
{
    class_id_t class_id;
    union_v_t value;
} union_t;
/* ----- END Union Type ----- */


/* ----- BEGIN Environment (lexical variables) ----- */
// environment_struct must be defined later
typedef struct environment_struct environment_t;
/* ----- END Environment (lexical variables) ----- */

struct environment_struct
{
    int l1_hx_res;
    float l1_magnify;
    int l1_hy_res;
    int l1_iter_max;
    int l2_inverted;
};
__device__ int _block_k_1_(environment_t *_env_, int j)
{
    float xx;
    int iter;
    float y;
    float x;
    float cy;
    float cx;
    int hy;
    int hx;
    int lex_iter_max = _env_->l1_iter_max;
    int lex_hy_res = _env_->l1_hy_res;
    float lex_magnify = _env_->l1_magnify;
    int lex_hx_res = _env_->l1_hx_res;
    {
        hx = ((j % lex_hx_res));
        hy = ((j / lex_hx_res));
        cx = ((((((((((((float) hx) / ((float) lex_hx_res))) - 0.5)) / lex_magnify)) * 3.0)) - 0.7));
        cy = ((((((((((float) hy) / ((float) lex_hy_res))) - 0.5)) / lex_magnify)) * 3.0));
        x = 0.0;
        y = 0.0;
        for (iter = 0; iter <= lex_iter_max; iter++)
        {
            {
                xx = ((((((x * x)) - ((y * y)))) + cx));
                y = ((((((2.0 * x)) * y)) + cy));
                x = xx;
                if (((((((x * x)) + ((y * y)))) > 100)))
                {
                    break;
                }
            }
        }
        iter--;
        {
            return (iter % 256);
        }
    }
}


__device__ int _block_k_2_(environment_t *_env_, int color)
{
    int lex_inverted = _env_->l2_inverted;
    if (((lex_inverted == 1)))
    {
        {
            {
                return (255 - color);
            }
        }
    }
    else
    {
        {
            return color;
        }
    }
}


__global__ void kernel(environment_t *_env_, int *_result_)
{
    int t_id = threadIdx.x + blockIdx.x * blockDim.x;

    if (t_id < 40000)
    {
        _result_[t_id] = _block_k_2_(_env_, _block_k_1_(_env_, threadIdx.x + blockIdx.x * blockDim.x));
    }
}


typedef struct result_t {
    int *result;
    int last_error;
} result_t;

#define checkErrorReturn(result_var, expr) \
if (result_var->last_error = expr) \
{\
    hipError_t error = hipGetLastError();\
    printf("!!! Cuda Failure %s:%d (%i): '%s'\n", __FILE__, __LINE__, expr, hipGetErrorString(error));\
    hipDeviceReset();\
    return result_var;\
}

extern "C" EXPORT result_t *launch_kernel(environment_t *host_env)
{
    // CUDA Initialization
    result_t *kernel_result = (result_t *) malloc(sizeof(result_t));

    hipError_t cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        kernel_result->last_error = -1;
        return kernel_result;
    }

    checkErrorReturn(kernel_result, hipFree(0));

    /* Modify host environment to contain device pointers addresses */
    

    /* Allocate device environment and copy over struct */
    environment_t *dev_env;
    checkErrorReturn(kernel_result, hipMalloc(&dev_env, sizeof(environment_t)));
    checkErrorReturn(kernel_result, hipMemcpy(dev_env, host_env, sizeof(environment_t), hipMemcpyHostToDevice));

    int *host_result = (int *) malloc(sizeof(int) * 40000);
    int *device_result;
    checkErrorReturn(kernel_result, hipMalloc(&device_result, sizeof(int) * 40000));
    
    dim3 dim_grid(157, 1, 1);
    dim3 dim_block(256, 1, 1);

    kernel<<<dim_grid, dim_block>>>(dev_env, device_result);

    checkErrorReturn(kernel_result, hipPeekAtLastError());
    checkErrorReturn(kernel_result, hipDeviceSynchronize());

    checkErrorReturn(kernel_result, hipMemcpy(host_result, device_result, sizeof(int) * 40000, hipMemcpyDeviceToHost));
    checkErrorReturn(kernel_result, hipFree(dev_env));

    kernel_result->result = host_result;
    return kernel_result;
}
