#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <chrono>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

using namespace std;


/* ----- BEGIN Shared Library Export ----- */
// taken from http://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux

#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT __declspec(dllexport)
    #define IMPORT __declspec(dllimport)
#elif defined(_GCC)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif
/* ----- END Shared Library Export ----- */

/* ----- BEGIN Class Type ----- */
typedef int obj_id_t;
typedef int class_id_t;
/* ----- END Class Type ----- */

/* ----- BEGIN Union Type ----- */
typedef union union_type_value {
    obj_id_t object_id;
    int int_;
    float float_;
    bool bool_;
} union_v_t;

typedef struct union_type_struct
{
    class_id_t class_id;
    union_v_t value;
} union_t;
/* ----- END Union Type ----- */


/* ----- BEGIN Environment (lexical variables) ----- */
// environment_struct must be defined later
typedef struct environment_struct environment_t;
/* ----- END Environment (lexical variables) ----- */

struct environment_struct
{
};
struct indexed_struct_2_lt_int_int_gt_t
{
    int field_0;
int field_1;
};





__device__ int _block_k_2_(environment_t *_env_, int index)
{
    
    {
        return (index % 25000);
    }
}


__device__ int _block_k_4_(environment_t *_env_, int index)
{
    
    {
        return (((index + 101)) % 25000);
    }
}


__device__ int _block_k_6_(environment_t *_env_, indexed_struct_2_lt_int_int_gt_t zipped)
{
    
    {
        return (zipped.field_0 * zipped.field_1);
    }
}


__global__ void kernel_3(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {



        
        _result_[_tid_] = _block_k_6_(_env_, ((indexed_struct_2_lt_int_int_gt_t) {_block_k_2_(_env_, _tid_), _block_k_4_(_env_, _tid_)}));
    }
}


typedef struct result_t {
    int *result;
    int last_error;

    uint64_t time_setup_cuda;
    uint64_t time_prepare_env;
    uint64_t time_kernel;
    uint64_t time_free_memory;
} result_t;

#define checkErrorReturn(result_var, expr) \
if (result_var->last_error = expr) \
{\
    hipError_t error = hipGetLastError();\
    printf("!!! Cuda Failure %s:%d (%i): '%s'\n", __FILE__, __LINE__, expr, hipGetErrorString(error));\
    hipDeviceReset();\
    return result_var;\
}

#define timeStartMeasure() start_time = chrono::high_resolution_clock::now();

#define timeReportMeasure(result_var, variable_name) \
end_time = chrono::high_resolution_clock::now(); \
result_var->time_##variable_name = chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

extern "C" EXPORT result_t *launch_kernel(environment_t *host_env)
{
    // Variables for measuring time
    chrono::high_resolution_clock::time_point start_time;
    chrono::high_resolution_clock::time_point end_time;

    // CUDA Initialization
    result_t *program_result = (result_t *) malloc(sizeof(result_t));

    timeStartMeasure();

    hipError_t cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        program_result->last_error = -1;
        return program_result;
    }

    checkErrorReturn(program_result, hipFree(0));

    timeReportMeasure(program_result, setup_cuda);


    /* Prepare environment */
timeStartMeasure();
    /* Allocate device environment and copy over struct */
    environment_t *dev_env;
    checkErrorReturn(program_result, hipMalloc(&dev_env, sizeof(environment_t)));
    checkErrorReturn(program_result, hipMemcpy(dev_env, host_env, sizeof(environment_t), hipMemcpyHostToDevice));

timeReportMeasure(program_result, prepare_env);

    /* Launch all kernels */
timeStartMeasure();
    int * _kernel_result_6;
    checkErrorReturn(program_result, hipMalloc(&_kernel_result_6, (sizeof(int) * 30000000)));
    int * _kernel_result_6_host = (int *) malloc((sizeof(int) * 30000000));
    kernel_3<<<29297, 1024>>>(dev_env, 30000000, _kernel_result_6);
    checkErrorReturn(program_result, hipPeekAtLastError());
    checkErrorReturn(program_result, hipDeviceSynchronize());

    checkErrorReturn(program_result, hipMemcpy(_kernel_result_6_host, _kernel_result_6, (sizeof(int) * 30000000), hipMemcpyDeviceToHost));

timeReportMeasure(program_result, kernel);

    /* Free device memory */
timeStartMeasure();
    checkErrorReturn(program_result, hipFree(_kernel_result_6));

timeReportMeasure(program_result, free_memory);

    program_result->result = _kernel_result_6_host;
    return program_result;
}
