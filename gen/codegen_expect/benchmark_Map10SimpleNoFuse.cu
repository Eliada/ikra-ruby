#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <chrono>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

using namespace std;


/* ----- BEGIN Shared Library Export ----- */
// taken from http://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux

#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT __declspec(dllexport)
    #define IMPORT __declspec(dllimport)
#elif defined(_GCC)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif
/* ----- END Shared Library Export ----- */

/* ----- BEGIN Class Type ----- */
typedef int obj_id_t;
typedef int class_id_t;
/* ----- END Class Type ----- */

/* ----- BEGIN Environment (lexical variables) ----- */
// environment_struct must be defined later
typedef struct environment_struct environment_t;
/* ----- END Environment (lexical variables) ----- */


/* ----- BEGIN Forward declarations ----- */
typedef struct result_t result_t;
/* ----- END Forward declarations ----- */

// Define program result variable. Also contains benchmark numbers.
result_t *program_result;

// Variables for measuring time
chrono::high_resolution_clock::time_point start_time;
chrono::high_resolution_clock::time_point end_time;

/* ----- BEGIN Macros ----- */
#define timeStartMeasure() start_time = chrono::high_resolution_clock::now();

#define timeReportMeasure(result_var, variable_name) \
end_time = chrono::high_resolution_clock::now(); \
result_var->time_##variable_name = result_var->time_##variable_name + chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();
/* ----- END Macros ----- */
struct indexed_struct_4_lt_int_int_int_int_gt_t
{
    int field_0;
int field_1;
int field_2;
int field_3;
};

/* ----- BEGIN Structs ----- */
struct variable_size_array_t {
    void *content;
    int size;

    variable_size_array_t(void *content_ = NULL, int size_ = 0) : content(content_), size(size_) { }; 

    static const variable_size_array_t error_return_value;
};

// error_return_value is used in case a host section terminates abnormally
const variable_size_array_t variable_size_array_t::error_return_value = 
    variable_size_array_t(NULL, 0);

/* ----- BEGIN Union Type ----- */
typedef union union_type_value {
    obj_id_t object_id;
    int int_;
    float float_;
    bool bool_;
    void *pointer;
    variable_size_array_t variable_size_array;

    __host__ __device__ union_type_value(int value) : int_(value) { };
    __host__ __device__ union_type_value(float value) : float_(value) { };
    __host__ __device__ union_type_value(bool value) : bool_(value) { };
    __host__ __device__ union_type_value(void *value) : pointer(value) { };
    __host__ __device__ union_type_value(variable_size_array_t value) : variable_size_array(value) { };

    __host__ __device__ static union_type_value from_object_id(obj_id_t value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_int(int value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_float(float value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_bool(bool value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_pointer(void *value)
    {
        return union_type_value(value);
    }

    __host__ __device__ static union_type_value from_variable_size_array_t(variable_size_array_t value)
    {
        return union_type_value(value);
    }
} union_v_t;

typedef struct union_type_struct
{
    class_id_t class_id;
    union_v_t value;

    __host__ __device__ union_type_struct(
        class_id_t class_id_ = 0, union_v_t value_ = union_v_t(0))
        : class_id(class_id_), value(value_) { };

    static const union_type_struct error_return_value;
} union_t;

// error_return_value is used in case a host section terminates abnormally
const union_type_struct union_t::error_return_value = union_type_struct(0, union_v_t(0));
/* ----- END Union Type ----- */

typedef struct result_t {
    variable_size_array_t result;
    int last_error;

    uint64_t time_setup_cuda;
    uint64_t time_prepare_env;
    uint64_t time_kernel;
    uint64_t time_free_memory;
    uint64_t time_transfer_memory;
    uint64_t time_allocate_memory;

    // Memory management
    vector<void*> *device_allocations;
} result_t;
/* ----- END Structs ----- */

struct array_command_1 {
    // Ikra::Symbolic::ArrayIndexCommand
    indexed_struct_4_lt_int_int_int_int_gt_t *result;
    __host__ __device__ array_command_1(indexed_struct_4_lt_int_int_int_int_gt_t *result = NULL) : result(result) { }
};
struct array_command_2 {
    // Ikra::Symbolic::ArrayCombineCommand
    int *result;
    array_command_1 *input_0;
    __host__ __device__ array_command_2(int *result = NULL, array_command_1 *input_0 = NULL) : result(result), input_0(input_0) { }
};
struct array_command_3 {
    // Ikra::Symbolic::FixedSizeArrayInHostSectionCommand
    int *result;
    variable_size_array_t input_0;
    __host__ __device__ array_command_3(int *result = NULL, variable_size_array_t input_0 = variable_size_array_t::error_return_value) : result(result), input_0(input_0) { }
    int size() { return input_0.size; }
};
struct array_command_5 {
    // Ikra::Symbolic::ArrayIndexCommand
    indexed_struct_4_lt_int_int_int_int_gt_t *result;
    __host__ __device__ array_command_5(indexed_struct_4_lt_int_int_int_int_gt_t *result = NULL) : result(result) { }
};
struct array_command_4 {
    // Ikra::Symbolic::ArrayCombineCommand
    int *result;
    array_command_3 *input_0;
    array_command_5 *input_1;
    __host__ __device__ array_command_4(int *result = NULL, array_command_3 *input_0 = NULL, array_command_5 *input_1 = NULL) : result(result), input_0(input_0), input_1(input_1) { }
};
struct environment_struct
{
};

// TODO: There should be a better to check if _block_k_2_ is already defined
#ifndef _block_k_2__func
#define _block_k_2__func
__device__ int _block_k_2_(environment_t *_env_, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    {
        return (((((((((7 + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_245(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_2_(_env_, ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_2_ is already defined
#ifndef _block_k_2__func
#define _block_k_2__func
__device__ int _block_k_2_(environment_t *_env_, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    {
        return (((((((((7 + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_247(environment_t *_env_, int _num_threads_, int *_result_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_2_(_env_, ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_249(environment_t *_env_, int _num_threads_, int *_result_, int *_array_251_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_251_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_252(environment_t *_env_, int _num_threads_, int *_result_, int *_array_254_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_254_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_255(environment_t *_env_, int _num_threads_, int *_result_, int *_array_257_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_257_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_258(environment_t *_env_, int _num_threads_, int *_result_, int *_array_260_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_260_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_261(environment_t *_env_, int _num_threads_, int *_result_, int *_array_263_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_263_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_264(environment_t *_env_, int _num_threads_, int *_result_, int *_array_266_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_266_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_267(environment_t *_env_, int _num_threads_, int *_result_, int *_array_269_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_269_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_270(environment_t *_env_, int _num_threads_, int *_result_, int *_array_272_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_272_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_273(environment_t *_env_, int _num_threads_, int *_result_, int *_array_275_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_275_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_276(environment_t *_env_, int _num_threads_, int *_result_, int *_array_278_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_278_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_279(environment_t *_env_, int _num_threads_, int *_result_, int *_array_281_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_281_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_282(environment_t *_env_, int _num_threads_, int *_result_, int *_array_284_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_284_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_285(environment_t *_env_, int _num_threads_, int *_result_, int *_array_287_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_287_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_288(environment_t *_env_, int _num_threads_, int *_result_, int *_array_290_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_290_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_291(environment_t *_env_, int _num_threads_, int *_result_, int *_array_293_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_293_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_294(environment_t *_env_, int _num_threads_, int *_result_, int *_array_296_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_296_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_297(environment_t *_env_, int _num_threads_, int *_result_, int *_array_299_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_299_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_300(environment_t *_env_, int _num_threads_, int *_result_, int *_array_302_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_302_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}



// TODO: There should be a better to check if _block_k_4_ is already defined
#ifndef _block_k_4__func
#define _block_k_4__func
__device__ int _block_k_4_(environment_t *_env_, int i, indexed_struct_4_lt_int_int_int_int_gt_t indices)
{
    
    
    
    {
        return (((((((((i + indices.field_0)) + indices.field_1)) + indices.field_2)) + indices.field_3)) % 1337);
    }
}

#endif


__global__ void kernel_303(environment_t *_env_, int _num_threads_, int *_result_, int *_array_305_)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    if (_tid_ < _num_threads_)
    {

        
        _result_[_tid_] = _block_k_4_(_env_, _array_305_[_tid_], ((indexed_struct_4_lt_int_int_int_int_gt_t) {_tid_ / 500000, (_tid_ / 1000) % 500, (_tid_ / 2) % 500, (_tid_ / 1) % 2}));
    }
}


#undef checkErrorReturn
#define checkErrorReturn(result_var, expr) \
if (result_var->last_error = expr) \
{\
    hipError_t error = hipGetLastError();\
    printf("!!! Cuda Failure %s:%d (%i): '%s'\n", __FILE__, __LINE__, expr, hipGetErrorString(error));\
    hipDeviceReset();\
    return variable_size_array_t::error_return_value;\
}

variable_size_array_t _host_section__(environment_t *host_env, environment_t *dev_env, result_t *program_result)
{
    array_command_2 * base = new array_command_2();
    array_command_4 * _ssa_var_base_11;
    array_command_3 * _ssa_var_base_10;
    array_command_3 * _ssa_var_base_9;
    array_command_3 * _ssa_var_base_8;
    array_command_3 * _ssa_var_base_7;
    array_command_3 * _ssa_var_base_6;
    array_command_3 * _ssa_var_base_5;
    array_command_3 * _ssa_var_base_4;
    array_command_3 * _ssa_var_base_3;
    array_command_3 * _ssa_var_base_2;
    array_command_3 * _ssa_var_base_1;
    {
        _ssa_var_base_1 = new array_command_3(NULL, ({
            // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]
        
            array_command_2 * cmd = base;
        
            if (cmd->result == 0) {
                    timeStartMeasure();
            int * _kernel_result_246;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_246, (sizeof(int) * 10000000)));
            program_result->device_allocations->push_back(_kernel_result_246);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_245<<<39063, 256>>>(dev_env, 10000000, _kernel_result_246);
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);
                cmd->result = _kernel_result_246;
        
                
            }
        
            variable_size_array_t((void *) cmd->result, 10000000);
        }));
        _ssa_var_base_2 = new array_command_3(NULL, ({
            // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [LVarReadNode: _ssa_var_base_1].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
        
            array_command_4 * cmd = new array_command_4(NULL, _ssa_var_base_1);
        
            if (cmd->result == 0) {
                    timeStartMeasure();
            int * _kernel_result_250;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_250, (sizeof(int) * 10000000)));
            program_result->device_allocations->push_back(_kernel_result_250);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_249<<<39063, 256>>>(dev_env, 10000000, _kernel_result_250, ((int *) ((int *) cmd->input_0->input_0.content)));
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);
                cmd->result = _kernel_result_250;
        
                
            }
        
            variable_size_array_t((void *) cmd->result, 10000000);
        }));
        _ssa_var_base_3 = new array_command_3(NULL, ({
            // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [LVarReadNode: _ssa_var_base_1].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
        
            array_command_4 * cmd = new array_command_4(NULL, _ssa_var_base_2);
        
            if (cmd->result == 0) {
                    timeStartMeasure();
            int * _kernel_result_256;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_256, (sizeof(int) * 10000000)));
            program_result->device_allocations->push_back(_kernel_result_256);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_255<<<39063, 256>>>(dev_env, 10000000, _kernel_result_256, ((int *) ((int *) cmd->input_0->input_0.content)));
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);
                cmd->result = _kernel_result_256;
        
                
            }
        
            variable_size_array_t((void *) cmd->result, 10000000);
        }));
        _ssa_var_base_4 = new array_command_3(NULL, ({
            // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [LVarReadNode: _ssa_var_base_1].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
        
            array_command_4 * cmd = new array_command_4(NULL, _ssa_var_base_3);
        
            if (cmd->result == 0) {
                    timeStartMeasure();
            int * _kernel_result_262;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_262, (sizeof(int) * 10000000)));
            program_result->device_allocations->push_back(_kernel_result_262);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_261<<<39063, 256>>>(dev_env, 10000000, _kernel_result_262, ((int *) ((int *) cmd->input_0->input_0.content)));
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);
                cmd->result = _kernel_result_262;
        
                
            }
        
            variable_size_array_t((void *) cmd->result, 10000000);
        }));
        _ssa_var_base_5 = new array_command_3(NULL, ({
            // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [LVarReadNode: _ssa_var_base_1].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
        
            array_command_4 * cmd = new array_command_4(NULL, _ssa_var_base_4);
        
            if (cmd->result == 0) {
                    timeStartMeasure();
            int * _kernel_result_268;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_268, (sizeof(int) * 10000000)));
            program_result->device_allocations->push_back(_kernel_result_268);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_267<<<39063, 256>>>(dev_env, 10000000, _kernel_result_268, ((int *) ((int *) cmd->input_0->input_0.content)));
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);
                cmd->result = _kernel_result_268;
        
                
            }
        
            variable_size_array_t((void *) cmd->result, 10000000);
        }));
        _ssa_var_base_6 = new array_command_3(NULL, ({
            // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [LVarReadNode: _ssa_var_base_1].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
        
            array_command_4 * cmd = new array_command_4(NULL, _ssa_var_base_5);
        
            if (cmd->result == 0) {
                    timeStartMeasure();
            int * _kernel_result_274;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_274, (sizeof(int) * 10000000)));
            program_result->device_allocations->push_back(_kernel_result_274);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_273<<<39063, 256>>>(dev_env, 10000000, _kernel_result_274, ((int *) ((int *) cmd->input_0->input_0.content)));
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);
                cmd->result = _kernel_result_274;
        
                
            }
        
            variable_size_array_t((void *) cmd->result, 10000000);
        }));
        _ssa_var_base_7 = new array_command_3(NULL, ({
            // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [LVarReadNode: _ssa_var_base_1].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
        
            array_command_4 * cmd = new array_command_4(NULL, _ssa_var_base_6);
        
            if (cmd->result == 0) {
                    timeStartMeasure();
            int * _kernel_result_280;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_280, (sizeof(int) * 10000000)));
            program_result->device_allocations->push_back(_kernel_result_280);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_279<<<39063, 256>>>(dev_env, 10000000, _kernel_result_280, ((int *) ((int *) cmd->input_0->input_0.content)));
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);
                cmd->result = _kernel_result_280;
        
                
            }
        
            variable_size_array_t((void *) cmd->result, 10000000);
        }));
        _ssa_var_base_8 = new array_command_3(NULL, ({
            // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [LVarReadNode: _ssa_var_base_1].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
        
            array_command_4 * cmd = new array_command_4(NULL, _ssa_var_base_7);
        
            if (cmd->result == 0) {
                    timeStartMeasure();
            int * _kernel_result_286;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_286, (sizeof(int) * 10000000)));
            program_result->device_allocations->push_back(_kernel_result_286);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_285<<<39063, 256>>>(dev_env, 10000000, _kernel_result_286, ((int *) ((int *) cmd->input_0->input_0.content)));
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);
                cmd->result = _kernel_result_286;
        
                
            }
        
            variable_size_array_t((void *) cmd->result, 10000000);
        }));
        _ssa_var_base_9 = new array_command_3(NULL, ({
            // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [LVarReadNode: _ssa_var_base_1].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
        
            array_command_4 * cmd = new array_command_4(NULL, _ssa_var_base_8);
        
            if (cmd->result == 0) {
                    timeStartMeasure();
            int * _kernel_result_292;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_292, (sizeof(int) * 10000000)));
            program_result->device_allocations->push_back(_kernel_result_292);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_291<<<39063, 256>>>(dev_env, 10000000, _kernel_result_292, ((int *) ((int *) cmd->input_0->input_0.content)));
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);
                cmd->result = _kernel_result_292;
        
                
            }
        
            variable_size_array_t((void *) cmd->result, 10000000);
        }));
        _ssa_var_base_10 = new array_command_3(NULL, ({
            // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [LVarReadNode: _ssa_var_base_1].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
        
            array_command_4 * cmd = new array_command_4(NULL, _ssa_var_base_9);
        
            if (cmd->result == 0) {
                    timeStartMeasure();
            int * _kernel_result_298;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_298, (sizeof(int) * 10000000)));
            program_result->device_allocations->push_back(_kernel_result_298);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_297<<<39063, 256>>>(dev_env, 10000000, _kernel_result_298, ((int *) ((int *) cmd->input_0->input_0.content)));
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);
                cmd->result = _kernel_result_298;
        
                
            }
        
            variable_size_array_t((void *) cmd->result, 10000000);
        }));
        _ssa_var_base_11 = new array_command_4(NULL, _ssa_var_base_10);
        return ({
            // [Ikra::Symbolic::ArrayCombineCommand, size = 10000000]: [SendNode: [LVarReadNode: _ssa_var_base_1].pmap([HashNode: {<:with_index> => [BeginNode: {<true>}]}])]
        
            array_command_4 * cmd = _ssa_var_base_11;
        
            if (cmd->result == 0) {
                    timeStartMeasure();
            int * _kernel_result_304;
            checkErrorReturn(program_result, hipMalloc(&_kernel_result_304, (sizeof(int) * 10000000)));
            program_result->device_allocations->push_back(_kernel_result_304);
            timeReportMeasure(program_result, allocate_memory);
            timeStartMeasure();
            kernel_303<<<39063, 256>>>(dev_env, 10000000, _kernel_result_304, ((int *) ((int *) cmd->input_0->input_0.content)));
            checkErrorReturn(program_result, hipPeekAtLastError());
            checkErrorReturn(program_result, hipDeviceSynchronize());
            timeReportMeasure(program_result, kernel);
                cmd->result = _kernel_result_304;
        
                
            }
        
            variable_size_array_t((void *) cmd->result, 10000000);
        });
    }
}

#undef checkErrorReturn
#define checkErrorReturn(result_var, expr) \
expr

extern "C" EXPORT result_t *launch_kernel(environment_t *host_env)
{
    // CUDA Initialization
    program_result = new result_t();
    program_result->device_allocations = new vector<void*>();

    timeStartMeasure();

    hipError_t cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        program_result->last_error = -1;
        return program_result;
    }

    checkErrorReturn(program_result, hipFree(0));

    timeReportMeasure(program_result, setup_cuda);


    /* Prepare environment */
        /* Allocate device environment and copy over struct */
    environment_t *dev_env;

    timeStartMeasure();
    checkErrorReturn(program_result, hipMalloc(&dev_env, sizeof(environment_t)));
    timeReportMeasure(program_result, allocate_memory);

    timeStartMeasure();
    checkErrorReturn(program_result, hipMemcpy(dev_env, host_env, sizeof(environment_t), hipMemcpyHostToDevice));
    timeReportMeasure(program_result, transfer_memory);
    


    /* Copy back memory and set pointer of result */
    program_result->result = ({
    variable_size_array_t device_array = _host_section__(host_env, dev_env, program_result);
    int * tmp_result = (int *) malloc(sizeof(int) * device_array.size);

    timeStartMeasure();
    checkErrorReturn(program_result, hipMemcpy(tmp_result, device_array.content, sizeof(int) * device_array.size, hipMemcpyDeviceToHost));
    timeReportMeasure(program_result, transfer_memory);

    variable_size_array_t((void *) tmp_result, device_array.size);
});

    /* Free device memory */
    timeStartMeasure();

    for (
        auto device_ptr = program_result->device_allocations->begin(); 
        device_ptr < program_result->device_allocations->end(); 
        device_ptr++)
    {
        checkErrorReturn(program_result, hipFree(*device_ptr));
    }

    delete program_result->device_allocations;

    timeReportMeasure(program_result, free_memory);

    return program_result;
}
